#include <hip/hip_runtime.h>
#include <stdio.h>


void cudasafe(int error, char* message, char* file, int line) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s : %i. In %s line %d\n", message, error, file, line);
        exit(-1);
    }
}


int main(int argc, char** argv) {
    int deviceCount;

    cudasafe(hipGetDeviceCount(&deviceCount), "GetDeviceCount", __FILE__, __LINE__);

    printf("Number of CUDA devices %d.\n", deviceCount);

    for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;

        cudasafe(hipGetDeviceProperties(&deviceProp, dev), "Get Device Properties", __FILE__, __LINE__);

        if (dev == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
                printf("No CUDA GPU has been detected\n");
                return -1;
            }
            else if (deviceCount == 1) {
                printf("There is 1 device supporting CUDA\n");
            }
            else {
                printf("There are %d devices supporting CUDA\n", deviceCount);
            }
        }

        printf("For device #%d\n", dev);
        printf("Device name:                %s\n", deviceProp.name);
        printf("Major revision number:      %d\n", deviceProp.major);
        printf("Minor revision Number:      %d\n", deviceProp.minor);
        printf("Total Global Memory:        %d\n", deviceProp.totalGlobalMem);
        printf("Total shared mem per block: %d\n", deviceProp.sharedMemPerBlock);
        printf("Total const mem size:       %d\n", deviceProp.totalConstMem);
        printf("Warp size:                  %d\n", deviceProp.warpSize);
        printf("Maximum block dimensions:   %d x %d x %d\n", deviceProp.maxThreadsDim[0], \
            deviceProp.maxThreadsDim[1], \
            deviceProp.maxThreadsDim[2]);

        printf("Maximum grid dimensions:    %d x %d x %d\n", deviceProp.maxGridSize[0], \
            deviceProp.maxGridSize[1], \
            deviceProp.maxGridSize[2]);
        printf("Clock Rate:                 %d\n", deviceProp.clockRate);
        printf("Number of muliprocessors:   %d\n", deviceProp.multiProcessorCount);

    }

    return 0;


}
