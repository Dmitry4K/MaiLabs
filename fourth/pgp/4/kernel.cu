#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include<thrust/extrema.h>
#include <thrust/device_vector.h>
#include<vector>
#include<cmath>
#include<algorithm>
#include <iomanip>
#include <random>
//#define BLOCKS_2D dim3(32,32)
//#define THREADS_2D dim3(32,32)
//#define BLOCKS_1D 256
//#define THREADS_1D 512
#define BLOCKS_2D dim3(8,8)
#define THREADS_2D dim3(8,8)
#define BLOCKS_1D 32
#define THREADS_1D 32
#define EPS 0.0000001

#define CUDA_ERROR_HANDLER(err) do { if (err != hipSuccess) { std::cout << hipGetErrorString(err) << std::endl; exit(1);} } while (0)
#define CEH(err) CUDA_ERROR_HANDLER(err)

#include<chrono>


namespace chrono_timer {
	template<class T = std::chrono::nanoseconds>
	class timer {
		std::chrono::steady_clock::time_point start_point = std::chrono::high_resolution_clock::now();
		std::chrono::steady_clock::time_point end_point = start_point;
		bool is_stoped = true;
	public:
		using time_type = T;
		void start() {
			start_point = std::chrono::high_resolution_clock::now();
			is_stoped = false;
		}
		void stop() {
			end_point = std::chrono::high_resolution_clock::now();
			is_stoped = true;
		}
		long long time() {
			return is_stoped ? std::chrono::duration_cast<time_type>(end_point - start_point).count() : std::chrono::duration_cast<time_type>(std::chrono::high_resolution_clock::now() - start_point).count();
		}
		long long measure(void f(void)) {
			this->start();
			f();
			return this->time();
		}
	};
	long long nanoseconds_time(void f(void)) {
		timer<> timer;
		timer.start();
		f();
		return timer.time();
	}

	template<class T>
	long long measure_time(void f(void)) {
		timer<T> timer;
		timer.start();
		f();
		return timer.time();
	}

	using timer_nns = timer <>;
	using timer_mls = timer < std::chrono::milliseconds >;
	using timer_mcs = timer < std::chrono::microseconds >;
	using timer_sec = timer < std::chrono::seconds >;
};
using namespace chrono_timer;

double drand(double b, double e)
{
	return b + (std::rand() % static_cast<unsigned int>((e - b) * 1000)) / 1000.;
}

__global__ void div_row_kernel(double* matrix, double lead, int row, int n) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;
	for (int i = row + id; i < n; i += offset) {
		matrix[row + i * n] /= lead;
	}
}

__global__ void sub_rows_kernel(double* matrix, int lead_row, int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	for (int i = lead_row + idx + 1; i < n; i += offsetx) {
		for (int j = lead_row + idy + 1; j < n; j += offsety) {
			matrix[j * n + i] -= matrix[lead_row*n + i] * matrix[lead_row + j*n];
		}
	}
}

__global__ void swap_rows_kernel(double* matrix, int first, int second, int off, int n) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;
	for (int i = off + id; i < n; i += offset) {
		double tmp = matrix[i * n + first];
		matrix[i * n + first] = matrix[i * n + second];
		matrix[i * n + second] = tmp;
	}
}

__global__ void set_zeros(double* matrix, int column, int n) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;
	for (int i = column + id + 1; i < n; i += offset) {
		matrix[i + column * n] = 0.0;
	}
}

struct comparator
{
	__host__ __device__ bool operator()(double a, double b)
	{
		return std::fabs(a) < std::fabs(b);
	}
};
double get_det(std::vector<double>& multipliers) {
	comparator comp;
	std::sort(multipliers.begin(), multipliers.end(), comp);
	double ans = 1.0;
	int begin = 0, end = (int)multipliers.size() - 1;

	while(begin <= end) {
		if (std::abs(ans) < EPS) {
			ans *= multipliers[end--];
		}
		else {
			ans *= multipliers[begin++];
		}
	}
	return ans;
}

int main() {
	int n;
	comparator comp;
	std::ios::sync_with_stdio(false);
	std::cin.tie(nullptr);
	std::cin >> n;
	double* matrix = new double[n * n];
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			//std::cin >> matrix[i + j * n];
			matrix[i + j * n] = drand(0, 1);
		}
	}
	std::cout << std::setprecision(10);
	std::cout.setf(std::ios::scientific);
	if (n == 1) {
		std::cout << matrix[0];
		return 0;
	}
	double* dev_matrix = nullptr;
	CEH(hipMalloc(&dev_matrix, sizeof(double) * n * n));
	CEH(hipMemcpy(dev_matrix, matrix, sizeof(double) * n * n, hipMemcpyHostToDevice));
	std::vector<double> multipliers(n);
	bool is_zero = false;
	thrust::device_ptr<double> begin;
	thrust::device_ptr<double> max_ptr;
	timer_nns timer;
	timer.start();
	for (int i = 0; i < n; ++i) {
		begin = thrust::device_pointer_cast(dev_matrix + i + i * n);
		double multiplier = *begin;
		if (std::abs(multiplier) <= EPS) {
			max_ptr = thrust::max_element(begin, begin + (n - i), comp);
			double max_el = *max_ptr;
			int max_index = max_ptr - begin;
			if (std::abs(max_el) <= EPS) {
				is_zero = true;
				break;
			}
			swap_rows_kernel << < BLOCKS_1D, THREADS_1D >> > (dev_matrix, i, max_index+i, i, n);
			multipliers[i] = -1.0 * max_el;
			multiplier = max_el;
		}
		else {
			multipliers[i] = multiplier;
		}
		div_row_kernel << < BLOCKS_1D, THREADS_1D >> > (dev_matrix, multiplier, i, n);
		sub_rows_kernel << <BLOCKS_2D, THREADS_2D >> > (dev_matrix, i, n);
		//set_zeros <<< BLOCKS_1D, THREADS_1D >> > (dev_matrix, i, n);
	}
	double determinant = is_zero ? 0 : get_det(multipliers);
	//std::cout << timer.time();
	std::cout << determinant;
	CEH(hipFree(dev_matrix));
	delete[] matrix;
	exit(0);
}
