#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>

#include<chrono>

namespace chrono_timer {
    template<class T = std::chrono::nanoseconds>
    class timer {
        std::chrono::steady_clock::time_point start_point = std::chrono::high_resolution_clock::now();
        std::chrono::steady_clock::time_point end_point = start_point;
        bool is_stoped = true;
    public:
        using time_type = T;
        void start() {
            start_point = std::chrono::high_resolution_clock::now();
            is_stoped = false;
        }
        void stop() {
            end_point = std::chrono::high_resolution_clock::now();
            is_stoped = true;
        }
        long long time() {
            return is_stoped ? std::chrono::duration_cast<time_type>(end_point - start_point).count() : std::chrono::duration_cast<time_type>(std::chrono::high_resolution_clock::now() - start_point).count();
        }
        long long measure(void f(void)) {
            this->start();
            f();
            return this->time();
        }
    };
    long long measure_time_in_nanoseconds(void f(void)) {
        timer<> timer;
        timer.start();
        f();
        return timer.time();
    }

    template<class T>
    long long measure_time(void f(void)) {
        timer<T> timer;
        timer.start();
        f();
        return timer.time();
    }

    using timer_nns = timer <>;
    using timer_mls = timer < std::chrono::milliseconds >;
    using timer_mcs = timer < std::chrono::microseconds >;
    using timer_sec = timer < std::chrono::seconds >;
};


#define CUDA_ERROR_HANDLER(err) do { if (err != hipSuccess) { std::cout << hipGetErrorString(err) << std::endl; exit(1);} } while (0)
#define CEH(err) CUDA_ERROR_HANDLER(err)
#define BLOCKS dim3(4, 4)
#define THREADS dim3(4, 4)

using namespace chrono_timer;

struct double3x3 {
    double x11, x12, x13, x21, x22, x23, x31, x32, x33;
};

__constant__ double3 AVG[32];
__constant__ double3x3 INV_COV[32];

__device__ __host__ double3 double3_from(uchar4 a) {
    return { (double)a.x, (double)a.y, (double)a.z };
}
__device__ __host__ double3 sum(double3 a, double3 b) {
    return { a.x + b.x, a.y + b.y, a.z + b.z };
}
__device__ __host__ double3x3 div(double3x3 a, double b) {
    return { a.x11 / b, a.x12 / b,a.x13 / b, a.x21 / b, a.x22 / b, a.x23 / b, a.x31 / b, a.x32 / b, a.x33 / b };
}
__device__ __host__ double3 div(double3 a, double b) {
    return { a.x / b, a.y / b, a.z / b };
}
__device__ __host__ double3 sub(double3 a, double3 b) {
    return { a.x - b.x, a.y - b.y, a.z - b.z };
}
__device__ __host__ double3x3 mult(double3 a, double3 b) {
    return {a.x * b.x, a.x * b.y, a.x * b.z,
            a.y * b.x, a.y * b.y, a.y * b.z,
            a.z * b.x, a.z * b.y, a.z * b.z };
}

__device__ __host__ double dot(double3 a, double3 b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}
__device__ __host__ double3 mult(double3 a, double3x3 b) {
    return { dot(a, {b.x11, b.x21,b.x31}), dot(a, {b.x12, b.x22,b.x32}), dot(a, {b.x13, b.x23,b.x33}) };
}


__device__ __host__ double3x3 sum(double3x3 a, double3x3 b) {
    return { a.x11 + b.x11, a.x12 + b.x12, a.x13 + b.x13,
             a.x21 + b.x21, a.x22 + b.x22, a.x23 + b.x23,
             a.x31 + b.x31, a.x32 + b.x32, a.x33 + b.x33};
}

__device__ __host__ double det(double3x3 a) {
    return a.x11 * a.x22 * a.x33 + a.x12 * a.x23 * a.x31 + a.x13 * a.x21 * a.x32 -
        a.x13 * a.x22 * a.x31 - a.x12 * a.x21 * a.x33 - a.x11 * a.x23 * a.x32;
}

__device__ __host__ double3x3 transpose(double3x3 a) {
    return { a.x11, a.x21, a.x31,
            a.x12, a.x22, a.x32,
            a.x13, a.x23, a.x33 };
}


__device__ __host__ double3x3 inv(double3x3 a) {
    a = transpose(a);
    double3x3 m = { (a.x22 * a.x33 - a.x32 * a.x23), -1.0* (a.x21 * a.x33 - a.x31 * a.x23), (a.x21 * a.x32 - a.x31 * a.x22),
                   -1.0 * (a.x12 * a.x33 - a.x32 * a.x13), (a.x11 * a.x33 - a.x31 * a.x13), -1.0* (a.x11 * a.x32 - a.x31 * a.x12),
                    (a.x12 * a.x23 - a.x22 * a.x13), -1.0 * (a.x11 * a.x23 - a.x21 * a.x13), (a.x11 * a.x22 - a.x21 * a.x12) };
    return div(m, det(a));
}

__device__ __host__ int argMax(uchar4 pixel, int nc) {
    double result, max;
    double3 tmp = sub(double3_from(pixel), AVG[0]);
    int clas = 0;
    max = -1.0 * dot(mult(tmp, INV_COV[0]), tmp);
    for (int i = 1; i < nc; ++i) {
        tmp = sub(double3_from(pixel), AVG[i]);
        result = -1.0 * dot(mult(tmp, INV_COV[i]), tmp);
        if (result > max) {
            max = result;
            clas = i;
        }
    }
    return clas;
}

void linearMahalanobisaMethod(uchar4* image, long long w, long long h, int nc) {
    for (long long y = 0; y < h; ++y) {
        for (long long x = 0; x < w; ++x) {
            //image[x + y * w].w 
        }
    }
}


__global__ void mahalanobisaMethod(uchar4* dev_image, long long w, long long h, int nc) {
    long long idx = blockDim.x * blockIdx.x + threadIdx.x;
    long long idy = blockDim.y * blockIdx.y + threadIdx.y;
    long long offsetx = blockDim.x * gridDim.x;
    long long offsety = blockDim.y * gridDim.y;

    for (long long y = idy; y < h; y += offsety) {
        for (long long x = idx; x < w; x += offsetx) {
            dev_image[x + y * w].w = argMax(dev_image[x + y * w], nc);
            /*int clas = argMax(dev_image[x + y * w], nc);
            dev_image[x + y * w] = make_uchar4(0, 0, 0, 255);
            if (clas == 0)
                dev_image[x + y * w].x = 240;
            if (clas == 1)
                dev_image[x + y * w].y = 240;
            if (clas == 2)
                dev_image[x + y * w].z = 240;*/
        }
    }
}

void print(double3x3 a) {
    std::cout << a.x11 << ' ' << a.x12 << ' ' << a.x13 << std::endl;
    std::cout << a.x21 << ' ' << a.x22 << ' ' << a.x23 << std::endl;
    std::cout << a.x31 << ' ' << a.x32 << ' ' << a.x33 << std::endl;
}

void print(double a) {
    std::cout << a << std::endl;
}

int main()
{
    std::string input_file_path, output_file_path;
    int nc, np;
    std::cin >> input_file_path >> output_file_path >> nc;
    std::vector<std::vector<int2>> classes(nc);
    for (auto& clas : classes) {
        std::cin >> np;
        clas.resize(np);
        for (auto& pair : clas) {
            std::cin >> pair.x >> pair.y;
        }
    }

    uchar4* image = nullptr;
    int w, h;

    std::ifstream input{ input_file_path , std::ios::in | std::ios::binary };
    if (input.is_open()) {
        input.read((char*)&w, sizeof(int));
        input.read((char*)&h, sizeof(int));
        image = new uchar4[(size_t)w * h];
        input.read((char*)image, sizeof(uchar4) * w * h);
        input.close();
    }
    else {
        exit(1);
    }

    std::vector<double3> avg(nc, { 0.0, 0.0, 0.0 });
    for (int j = 0; j < avg.size(); ++j) {
        double3 s = { 0.0, 0.0, 0.0 };
        auto& clas = classes[j];
        int np = (int)clas.size();
        for (auto& el : clas) {
            uchar4 ps = image[el.y * w + el.x];
            s = sum(s, double3_from(ps));
        }
        avg[j] = div(s, np);
    }


    std::vector<double3x3> inv_cov(nc, { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 });
    for (int j = 0; j < inv_cov.size(); ++j) {
        double3x3 s = { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 };
        auto& clas = classes[j];
        int np = (int) clas.size();
        for (auto& el : clas) {
            uchar4 ps = image[el.y * w + el.x];
            double3 tmp = sub(double3_from(ps), avg[j]);
            s = sum(s, mult(tmp, tmp));
        }
        inv_cov[j] = inv(div(s, np - 1));
    }
    timer_nns timer;
    CEH(hipMemcpyToSymbol(HIP_SYMBOL(AVG), avg.data(), sizeof(double3) * avg.size()));
    CEH(hipMemcpyToSymbol(HIP_SYMBOL(INV_COV), inv_cov.data(), sizeof(double3x3) * inv_cov.size()));
    uchar4* dev_image = nullptr;
    CEH(hipMalloc(&dev_image, sizeof(uchar4) * h * w));
    CEH(hipMemcpy(dev_image, image, sizeof(uchar4) * h * w, hipMemcpyHostToDevice));
    timer.start();
    mahalanobisaMethod << < BLOCKS, THREADS >> > (dev_image, w, h, nc);
    std::cout << "Cuda time: " << timer.time() << std::endl;
    //timer.start();
    //linearMahalanobisaMethod(image, w, h, nc);
    //std::cout << "Linear time: " << timer.time() << std::endl;
    CEH(hipMemcpy(image, dev_image, sizeof(uchar4) * h * w, hipMemcpyDeviceToHost));
    for (long long y = 0; y < h; y ++) {
        for (long long x = 0; x < w; x ++) {
            
            //*
            int clas = image[x + y * w].w;
                image[x + y * w] = make_uchar4(0, 0, 0, 255);
            if (clas == 0)
                image[x + y * w].x = 240;
            if (clas == 1)
                image[x + y * w].y = 240;//*/
        }
    }
    std::ofstream output{ output_file_path ,std::ios::out | std::ios::binary };
    if (output.is_open()) {
        output.write((char*)&w, sizeof(int));
        output.write((char*)&h, sizeof(int));
        output.write((char*)image, sizeof(uchar4) * w * h);
        output.close();
    }
    else {
        exit(1);
    }
    CEH(hipFree(dev_image));
    delete[] image;
    return 0;
}
