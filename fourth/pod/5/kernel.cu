#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime_api.h>
#include <>
#include <thrust/scan.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <stack>
#include <iostream>

#define FLT_MAX 3.402823466e+38F
#define FLT_MIN 1.175494351e-38F

#define REDUCE_BLOCKS 256
#define REDUCE_TRHREADS_COUNT 256
#define BITONIC_SORT_BLOCKS_COUNT 512
#define BITONIC_SORT_THREADS_COUNT 512
#define RESET_BLOCKS_COUNT 32
#define RESET_THREADS_COUNT 32
#define COPY_BLOCKS_COUNT 64
#define THREADS_COPY_COUNT 32
#define COUNTING_SORT_BLOCKS 64
#define COUNTING_SORT_THREADS 32
#define POCKET_SIZE 1024


struct block_view {
    int start, end;
    __host__ __device__ int length() {
        return end - start + 1;
    }
};

__device__ void load_block_from_shared_to_global(
    volatile float* shared, volatile float* global, int tid, block_view view
) {
    if (tid < view.length()) global[view.start + tid] = shared[tid];
    if (tid + blockDim.x < view.length()) global[view.start + tid + blockDim.x] = shared[tid + blockDim.x];
    __syncthreads();
}

__device__ void load_block_from_global_to_shared(
    volatile float* shared, volatile float* global, int tid, block_view view
) {
    shared[tid] = (tid < view.length() ? global[view.start + tid] : FLT_MAX);
    shared[tid + blockDim.x] = (tid + blockDim.x < view.length() ? global[view.start + tid + blockDim.x] : FLT_MAX);
    __syncthreads();
}

__device__ void bytonic_sort_for_shared(volatile float* shared) { 
    for (int bitonic_length = 2; bitonic_length <= 2 * blockDim.x; bitonic_length *= 2) { 
        for (int block_size = bitonic_length; block_size >= 2; block_size /= 2) { 

            int filter_offset = block_size / 2;
            int block_offset = threadIdx.x / filter_offset * block_size;
            int filter_index = threadIdx.x % filter_offset;
            int first = block_offset + filter_index;
            int second = first + filter_offset;

            float min = fmin(shared[first], shared[second]);
            float max = fmax(shared[first], shared[second]);

            if (first & bitonic_length) {
                shared[first] = max;
                shared[second] = min;
            }
            else {
                shared[first] = min;
                shared[second] = max;
            }
            __syncthreads();
        }
    }
}

__global__ void bitonic_sort_global(float* global, block_view* blocks, int blocks_size) {
    extern __shared__ float shared[];
    for (int block_id = blockIdx.x; block_id < blocks_size; block_id += gridDim.x) {
        load_block_from_global_to_shared(shared, global, threadIdx.x, blocks[block_id]);
        bytonic_sort_for_shared(shared);
        load_block_from_shared_to_global(shared, global, threadIdx.x, blocks[block_id]);
    }
}

__device__ void reduce_for_mins_small(volatile float* mins) {
    for (int offset = 32; offset >= 1; offset /= 2)
        mins[threadIdx.x] = fmin(mins[threadIdx.x], mins[threadIdx.x + offset]);
}

__device__ void reduce_for_maxs_small(volatile float* maxs) {
    for (int offset = 32; offset >= 1; offset /= 2) 
        maxs[threadIdx.x] = fmax(maxs[threadIdx.x], maxs[threadIdx.x + offset]);
}

__device__ void reduce_for_mins_big(volatile float* shared_mins, volatile float* g_mem, int n) { 
    int idx = 2 * blockIdx.x * blockDim.x + threadIdx.x; 
    int offset = 2 * blockDim.x * gridDim.x; 
    shared_mins[threadIdx.x] = FLT_MAX; 
    for (; (idx + blockDim.x) < n; idx += offset) { 
        shared_mins[threadIdx.x] = fmin(fmin(shared_mins[threadIdx.x], g_mem[idx]), g_mem[idx + blockDim.x]); 
    }

    if (idx < n) {
        shared_mins[threadIdx.x] = fmin(shared_mins[threadIdx.x], g_mem[idx]);
    }
    __syncthreads();
}

__device__ void reduce_for_maxs_big(
    volatile float* shared_maxs, volatile float* g_mem, int n
) {
    int idx = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    int offset = 2 * blockDim.x * gridDim.x;
    shared_maxs[threadIdx.x] = FLT_MIN;
    for (; (idx + blockDim.x) < n; idx += offset) {
        shared_maxs[threadIdx.x] = fmax(fmax(shared_maxs[threadIdx.x], g_mem[idx]), g_mem[idx + blockDim.x]);
    }
    if (idx < n) {
        shared_maxs[threadIdx.x] = fmax(shared_maxs[threadIdx.x], g_mem[idx]);
    }
    __syncthreads();
}

__global__ void block_reduce_mins_kernel(float* g_mem, float* block_mins, int n) {
    extern __shared__ float s_memory[];
    reduce_for_mins_big(s_memory, g_mem, n);

    for (int offset = blockDim.x / 2; offset >= 64; offset /= 2) {
        if (threadIdx.x < offset)
            s_memory[threadIdx.x] = fmin(s_memory[threadIdx.x], s_memory[threadIdx.x + offset]);
        __syncthreads();
    }
    if (threadIdx.x < 32) reduce_for_mins_small(s_memory);

    if (threadIdx.x == 0) block_mins[blockIdx.x] = s_memory[0];
}

__global__ void block_reduce_maxs_kernel(float* g_mem, float* block_maxs, int n) {
    extern __shared__ float s_memory[];
    reduce_for_maxs_big(s_memory, g_mem, n);

    for (int offset = blockDim.x / 2; offset >= 64; offset /= 2) {
        if (threadIdx.x < offset)
            s_memory[threadIdx.x] = fmax(s_memory[threadIdx.x], s_memory[threadIdx.x + offset]);
        __syncthreads();
    }
    if (threadIdx.x < 32) reduce_for_maxs_small(s_memory);
    if (threadIdx.x == 0) block_maxs[blockIdx.x] = s_memory[0];
}


__global__ void hist_kernel(
    float* src, float* dst, int n,
    int* hist, int hist_size,
    float minimum, float maximum) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;
    for (int i = idx;  i < n; i += offset) {
        int mult_coef = ((src[i] - minimum) / (maximum - minimum));
        int pocket = hist_size - 1;
        if (mult_coef > 1) {
            pocket *= mult_coef;
        }
        atomicAdd(hist + pocket, 1u);
        dst[i] = src[i];
    }
}


__global__ void counting_sort(
    float* src, float* dst, int n,
    int* scan, int scan_size,
    float minimum, float maximum
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for (int i = tid; i < n; i += offset) {
        int pocket = scan_size - 1;
        int mult_coef = (src[i] - minimum) / (maximum-minimum);
        if (mult_coef > 1) pocket *= mult_coef;
        dst[atomicAdd(scan + pocket, 1)] = src[i];
    }
}


__global__ void set_nulls(int* arr, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for (int i = idx; i < size; i += offset) arr[i] = 0; 
}


float find_min(float* arr, int n) { 
    float min = arr[0]; 
    for (int i = 1; i < n; ++i) min = std::min(min, arr[i]); 
    return min; 
}

float find_max(float* arr, int n) {
    float max = arr[0];
    for (int i = 1; i < n; ++i) max = std::max(max, arr[i]);
    return max;
}

const int partition_effort = 2;
#define IOSTREAM

int main() {
    std::ios::sync_with_stdio(false);
    int arr_size;
#ifndef IOSTREAM
    fread(&arr_size, sizeof(int), 1, stdin);
#else
    std::cin >> arr_size;
#endif

    float* arr = (float*)malloc(arr_size * sizeof(float));

#ifndef IOSTREAM
    fread(arr, sizeof(float), arr_size, stdin);
    for (int i = 0; i < arr_size; ++i) std::cerr << arr[i] << ' ';
#else
    for (int i = 0; i < arr_size; ++i) arr[i] = float(arr_size - i) + 0.5;
#endif
    const int pockets_max_count = ((arr_size + POCKET_SIZE - 1) / POCKET_SIZE) * partition_effort;

    float* mins = (float*)malloc(REDUCE_BLOCKS * sizeof(float));
    float* maxs = (float*)malloc(REDUCE_BLOCKS * sizeof(float));
    int*   hist = (int*)  malloc(pockets_max_count * sizeof(int));
    int*   scan = (int*)  malloc(pockets_max_count * sizeof(int));

    float* dev_mins = nullptr;
    float* dev_maxs = nullptr;
    float* dev_array = nullptr;
    float* dev_counting_sort_buffer = nullptr;
    int* dev_hist_for_trust = nullptr;
    int* dev_scan = nullptr;

    fprintf(stderr, "%d\n", arr_size);
    hipMalloc(&dev_mins, REDUCE_BLOCKS * sizeof(float));
    hipMalloc(&dev_maxs, REDUCE_BLOCKS * sizeof(float));
    hipMalloc(&dev_array, arr_size * sizeof(float));
    hipMalloc(&dev_counting_sort_buffer, arr_size * sizeof(float));
    hipMalloc(&dev_hist_for_trust, pockets_max_count * sizeof(int));
    hipMalloc(&dev_scan, pockets_max_count * sizeof(int));
    
    thrust::device_ptr<int> dev_trust_hist = thrust::device_pointer_cast(dev_hist_for_trust);
    thrust::device_ptr<int> thrust_scan = thrust::device_pointer_cast(dev_scan);

    hipMemcpy(dev_array, arr, arr_size * sizeof(float), hipMemcpyHostToDevice);

    std::stack<block_view> pool;
    std::vector<block_view> pockets;

    if (arr_size > 0) {
        pool.push({ 0, arr_size-1 });
    }

    while (!pool.empty()) {
        block_view view = pool.top();
        pool.pop();
        if (view.length() <= POCKET_SIZE) {
            pockets.push_back(view);
            continue;
        }
        block_reduce_mins_kernel << <REDUCE_BLOCKS, REDUCE_TRHREADS_COUNT,
            2 * REDUCE_TRHREADS_COUNT * sizeof(float) >> > (dev_array + view.start, dev_mins, view.length());
        block_reduce_maxs_kernel << <REDUCE_BLOCKS, REDUCE_TRHREADS_COUNT,
            2 * REDUCE_TRHREADS_COUNT * sizeof(float) >> > (dev_array + view.start, dev_maxs, view.length());
        hipMemcpy(mins, dev_mins, REDUCE_BLOCKS
            * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(maxs, dev_maxs, REDUCE_BLOCKS
            * sizeof(float), hipMemcpyDeviceToHost);
        float min = find_min(mins, REDUCE_BLOCKS);
        float max = find_max(maxs, REDUCE_BLOCKS);

        if (min != max) {
            int pockets_count = ((view.length() + POCKET_SIZE - 1) / POCKET_SIZE) * partition_effort;
            set_nulls << <RESET_BLOCKS_COUNT, RESET_THREADS_COUNT >> > (dev_hist_for_trust, pockets_count);
            hipDeviceSynchronize();

            hist_kernel << <COPY_BLOCKS_COUNT, THREADS_COPY_COUNT >> > (dev_array + view.start,
                dev_counting_sort_buffer, view.length(), dev_hist_for_trust, pockets_count, min, max);
            hipDeviceSynchronize();

            thrust::exclusive_scan(dev_trust_hist, dev_trust_hist + pockets_count, thrust_scan);
            hipMemcpy(hist, dev_hist_for_trust, pockets_count * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(scan, dev_scan, pockets_count * sizeof(int), hipMemcpyDeviceToHost);

            counting_sort << <COUNTING_SORT_BLOCKS, COUNTING_SORT_THREADS >> > (
                dev_counting_sort_buffer, dev_array + view.start, view.length(),
                dev_scan, pockets_count, min, max);
            hipDeviceSynchronize();

            for (int i = 0; i < pockets_count; ++i) {
                if (hist[i] != 0) {
                    //fprintf(stderr, "%d %d\n", (int)view.start + scan[i], (int)view.start + scan[i] + hist[i] - 1);
                    pool.push({ view.start + scan[i], view.start + scan[i] + hist[i] - 1 });
                }
            }
        }
        //fprintf(stderr, "%d\n", (int)pool.size());
    }

    hipFree(dev_hist_for_trust);
    hipFree(dev_scan);
    hipFree(dev_counting_sort_buffer);
    hipFree(dev_mins);
    hipFree(dev_maxs);
    free(maxs);
    free(mins);
    free(scan);
    free(hist);

    block_view* dev_pockets = nullptr;
    hipMalloc(&dev_pockets, pockets.size() *  sizeof(block_view));
    hipMemcpy(dev_pockets, pockets.data(), pockets.size() * sizeof(block_view), hipMemcpyHostToDevice);

    bitonic_sort_global << <BITONIC_SORT_BLOCKS_COUNT, BITONIC_SORT_THREADS_COUNT,
        2 * BITONIC_SORT_THREADS_COUNT * sizeof(float) >> > (
            dev_array, dev_pockets, pockets.size());

    hipMemcpy(arr, dev_array, arr_size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dev_pockets);

#ifndef IOSTREAM
    fwrite((char*)(arr), sizeof(float), arr_size, stdout);
#else
    for (int i = 0; i < arr_size; ++i) std::cout << arr[i] << ' ';
#endif
    free(arr);
    return 0;
}
