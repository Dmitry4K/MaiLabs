#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cmath>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
//#include<chrono>


//#define PRE_FILE_READING

#define BLOCKS dim3(16, 16)
#define THREADS dim3(16, 16)

#define CUDA_ERROR_HANDLER(err) do { if (err != hipSuccess) { std::cout << hipGetErrorString(err) << std::endl; exit(1);} } while (0)
#define CEH(err) CUDA_ERROR_HANDLER(err)

/*
namespace chrono_timer {
	template<class T = std::chrono::nanoseconds>
	class timer {
		std::chrono::steady_clock::time_point start_point = std::chrono::high_resolution_clock::now();
		std::chrono::steady_clock::time_point end_point = start_point;
		bool is_stoped = true;
	public:
		using time_type = T;
		void start() {
			start_point = std::chrono::high_resolution_clock::now();
			is_stoped = false;
		}
		void stop() {
			end_point = std::chrono::high_resolution_clock::now();
			is_stoped = true;
		}
		long long time() {
			return is_stoped ? std::chrono::duration_cast<time_type>(end_point - start_point).count() : std::chrono::duration_cast<time_type>(std::chrono::high_resolution_clock::now() - start_point).count();
		}
		long long measure(void f(void)) {
			this->start();
			f();
			return this->time();
		}
	};
	long long nanoseconds_time(void f(void)) {
		timer<> timer;
		timer.start();
		f();
		return timer.time();
	}

	template<class T>
	long long measure_time(void f(void)) {
		timer<T> timer;
		timer.start();
		f();
		return timer.time();
	}

	using timer_nns = timer <>;
	using timer_mls = timer < std::chrono::milliseconds >;
	using timer_mcs = timer < std::chrono::microseconds >;
	using timer_sec = timer < std::chrono::seconds >;
	using timer_min = timer < std::chrono::minutes >;
};*/

const float PI = 3.141592653589f;

const float SPEED = 0.02f;
const float OFFSET = 0.3f;

namespace COLORS {
	uchar4 RED = uchar4{ 255, 0, 0 };
	uchar4 BLUE = uchar4{ 0, 0, 255 };
	uchar4 GREEN = uchar4{ 0, 255, 0 };
	uchar4 PINK = uchar4{ 225, 0 ,255 };
	uchar4 YELLOW = uchar4{ 255, 255, 0 };
	uchar4 CYAN = uchar4{ 0, 255, 255 };
	uchar4 ORANGE = uchar4{ 240, 116,39 };
	uchar4 PURPLE = uchar4{ 148, 87, 235 };
}

__device__ __host__ float3 operator-(const float3& a, const float3& b) { return make_float3(a.x - b.x, a.y - b.y, a.z - b.z); }
__device__ __host__ float3 operator+(const float3& a, const float3& b) { return make_float3(a.x + b.x, a.y + b.y, a.z + b.z); }
__device__ __host__ float3 operator*(const float3& a, const float b) { return make_float3(a.x * b, a.y * b, a.z * b); }
__device__ __host__ float3 operator*(const float3& a, const float3& b) { return make_float3(a.x * b.x, a.y * b.y, a.z * b.z); }
std::ostream& operator<<(std::ostream& out, const float3& a) { return (out << a.x << ' ' << a.y << ' ' << a.z); }
std::istream& operator>>(std::istream& in, float3& a) { return (in >> a.x >> a.y >> a.z); }


__device__ __host__ float dot(float3 a, float3 b) { return a.x * b.x + a.y * b.y + a.z * b.z; }
__device__ __host__ float3 prod(float3 a, float3 b) { return { a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x }; }
__device__ __host__ float3 norm(float3 v) { float l = sqrtf(dot(v, v)); return { v.x / l, v.y / l, v.z / l }; }
__device__ __host__ float len(float3 a) { return sqrtf(a.x * a.x + a.y * a.y + a.z * a.z); }
__device__ __host__ float3 mult(float3 a, float3 b, float3 c, float3 v) {
	return { a.x * v.x + b.x * v.y + c.x * v.z,
				a.y * v.x + b.y * v.y + c.y * v.z,
				a.z * v.x + b.z * v.y + c.z * v.z };
}

using uchar = unsigned char;

struct key_config {
	key_config(int argc, char* argv[]) {

		if (argc == 1) {
			is_gpu = true; return;
		}

		if (strcmp(argv[1], "--gpu") == 0) is_gpu = true;

		else if (strcmp(argv[1], "--cpu") == 0) {
			is_cpu = true;
			is_gpu = false;
		}
		else if (strcmp(argv[1], "--default") == 0) {
			is_default = true;
			is_gpu = false;
		}
		else {
			std::cout << "Bad key: " << argv[1] << std::endl;
		}
		if (((int)is_gpu + (int)is_cpu + (int)is_default) != 1) {
			std::cout << "Choose only one type of execution: on CPU (--cpu) or on GPU (--gpu) or DEFAULT (--default)" << std::endl;
			exit(0);
		}
	}
	bool is_gpu = false, is_cpu = false, is_default = false;
};
struct camera_settings {
	int width;
	int height;
	int scaled_width;
	int scaled_height;
	long long total_pixel_count;
	long long total_scaled_pixel_count;
	float angle;
	void init(int sqrt_rays_per_pixel) {
		scaled_height = height * sqrt_rays_per_pixel;
		scaled_width = width * sqrt_rays_per_pixel;
		total_pixel_count = (long long)height * width;
		total_scaled_pixel_count = (long long)scaled_height * scaled_width;
	}
} CAMERA_SETTINGS;
std::istream& operator>>(std::istream& in, camera_settings& cs) {
	return (in >> cs.width >> cs.height >> cs.angle);
}
struct cylindrical_coordinates {
private:
	float r, A_r, A_z, omega_r, omega_z, omega_phi, p_r, p_z, z, phi;
public:
	float rByT(float t) {
		return r + A_r * sinf(omega_r * t + p_r);
	}
	float zByT(float t) {
		return z + A_z * sinf(omega_z * t + p_z);
	}
	float phiByT(float t) {
		return phi + omega_phi * t;
	}
	float3 vecByT(float t) {
		float r = rByT(t);
		float phi = phiByT(t);
		float z = zByT(t);
		return { r * cosf(phi), r * sinf(phi), z };
	}
	friend std::istream& operator>>(std::istream& in, cylindrical_coordinates& cc);
};
std::istream& operator>>(std::istream& in, cylindrical_coordinates& cc) {
	return (in >> cc.r >> cc.z >> cc.phi >> cc.A_r >> cc.A_z >> cc.omega_r >> cc.omega_z >> cc.omega_phi >> cc.p_r >> cc.p_z);
}
struct camera_movement_settings {
	cylindrical_coordinates positionMovement;
	cylindrical_coordinates viewMovement;
	friend std::istream& operator>>(std::istream& in, camera_movement_settings& cms);
} CAMERA_MOVEMENT_SETTINGS;
std::istream& operator>>(std::istream& in, camera_movement_settings& cms) {
	return (in >> cms.positionMovement >> cms.viewMovement);
}
struct floor_settings {
	float3 point1, point2, point3, point4, color;
	std::string texturePath;
	float reflection;
} FLOOR;
std::istream& operator>>(std::istream& in, floor_settings& fs) {
	return (in >> fs.point1 >> fs.point2 >> fs.point3 >> fs.point4 >> fs.texturePath >> fs.color >> fs.reflection);
}
struct light {
	float3 position, color;
	float intensity = 1.5;
};
std::istream& operator>>(std::istream& in, light& l) {
	return (in >> l.position >> l.color);
}
struct base_figure {
	float3 position, color;
	float radius, reflection, transparency; // отражение прозрачность
	int countOfLights;
	bool isLight = false;
	static base_figure fromLight(const light& l) {
		auto light_cube = base_figure();
		light_cube.position = l.position;
		light_cube.color = l.color;
		light_cube.radius = 0.01f;
		light_cube.isLight = true;
		return light_cube;
	}
};
std::istream& operator>>(std::istream& in, base_figure& f) {
	return (in >> f.position >> f.color >> f.radius >> f.reflection >> f.transparency >> f.countOfLights);
}


int FRAMES_COUNT;
struct trig {
	float3 a;
	float3 b;
	float3 c;
	uchar4 color;
	bool isLight = false;
	float reflection;
	trig(float3 a_, float3 b_, float3 c_, uchar4 color_, bool isLight_, float reflection_) {
		a = a_;
		b = b_;
		c = c_;
		color = color_;
		isLight = isLight_;
		reflection = reflection_;
	}
};

std::vector<trig> TRIGS;
std::string OUTPUT_RENDERED_IMAGE_PATH;
base_figure FIGURE_1;
base_figure FIGURE_2;
base_figure FIGURE_3;
int LIGHTS_COUNT;
std::vector<light> LIGHTS;
int RECURSIVE_DEPTH;
int SQRT_OF_LIGHTS_COUNT_PER_PIXEL;

struct loading_cash {
	float prev_percent = 0.0;
	void init() {
		prev_percent = 0.0;
	}
	void print_label_if_percent_significantly_increased(float current_percent, int frame_num, long long pixel_num) {
		if (std::abs(current_percent - prev_percent) > 5.0 || prev_percent == 0.0) {
			prev_percent = current_percent;
			std::cout << "\rRendering " << frame_num + 1 << " of " << FRAMES_COUNT << " each containing ";
			std::cout << CAMERA_SETTINGS.total_scaled_pixel_count << " pixels.. [";
			int full_tens = (int)current_percent / 10;
			int last_tens = 10 - full_tens;
			int remainder = (int)current_percent % 10;
			for (int i = 0; i < full_tens; ++i) {
				std::cout << "*";
			}
			if (remainder != 0) {
				--last_tens;
				std::cout << "+";
			}
			for (int i = 0; i < last_tens; ++i) {
				std::cout << ".";
			}
			std::cout << "]" << std::setw(3) << (int)current_percent << "%";
		}
	}
} LOADING_CASH;

void input() {
#ifndef PRE_FILE_READING
	std::istream& in = std::cin;
#else
	std::ifstream in = std::ifstream("C:\\dev\\CUDA\\KP\\KP\\1\\test1.txt");
	if (!in.is_open()) {
		std::cout << "BAD FILE";
		exit(0);
	}
#endif
	in >> FRAMES_COUNT >> OUTPUT_RENDERED_IMAGE_PATH >> CAMERA_SETTINGS;
	in >> CAMERA_MOVEMENT_SETTINGS;
	in >> FIGURE_1 >> FIGURE_2 >> FIGURE_3;
	in >> FLOOR >> LIGHTS_COUNT;
	for (int i = 0; i < LIGHTS_COUNT; ++i) {
		light l;
		in >> l;
		LIGHTS.push_back(l);
	}
	in >> RECURSIVE_DEPTH >> SQRT_OF_LIGHTS_COUNT_PER_PIXEL;
}
void init() {
	CAMERA_SETTINGS.init(SQRT_OF_LIGHTS_COUNT_PER_PIXEL);
}
void default_calc() {
	std::cout << "100" << std::endl;
	std::cout << "./img_ % d.data 1280 720 40" << std::endl;
	std::cout << "9.0 3.0 0.0 0.0 2.0 0.0 2.0 1.2 1.0 0.0" << std::endl;
	std::cout << "0.0 0.0 0.0 0.0 0.5 3.0 5.0 0.1 3.0 0.2" << std::endl;
	std::cout << "-1.0 0.15 -0.5 0.5 0.5 0.4 0.1 0.8 0.2 5" << std::endl;
	std::cout << "-1.2 -1.5 -0.0 1.0 0.0 0.0 0.4 0.9 0.1 5" << std::endl;
	std::cout << "0.1 0.0 0.0 0.0 0.7 0.7 0.7 0.7 0.3 2" << std::endl;
	std::cout << "-5.0 -5.0 -1.0 -5.0 5.0 -1.0 5.0 5.0 -1.0 5.0 -5.0 -1.0 ~/floor.data 1.0 1.0 0.0 0.5" << std::endl;
	std::cout << "4" << std::endl;
	std::cout << "1.9 1.9 0.3 1.0 1.0 1.0" << std::endl;
	std::cout << "-0.5 -1.0 0.7	1.0 0.6 1.0" << std::endl;
	std::cout << "1.5 -0.8 0.4 1.0 1.0 0.0" << std::endl;
	std::cout << "-1.9 1.9 0.0 0.7 1.0 1.0" << std::endl;
	std::cout << "1 1" << std::endl;
}

__host__ std::pair<int, float> find_crossing(float3 pos, float3 dir, int igTrigIndex = -1, bool ignoreLights = false) {
	int k_min = -1;
	float ts_min;
	for (int k = 0; k < TRIGS.size(); k++) {
		if (k == igTrigIndex) {
			continue;
		}
		trig& curTrig = TRIGS[k];
		if (curTrig.isLight && ignoreLights) {
			continue;
		}
		float3 e1 = curTrig.b - curTrig.a;
		float3 e2 = curTrig.c - curTrig.a;
		float3 p = prod(dir, e2);
		float div = dot(p, e1);
		if (fabs(div) < 1e-10)
			continue;
		float3 t = pos - curTrig.a;
		float u = dot(p, t) / div;
		if (u < 0.0 || u > 1.0)
			continue;
		float3 q = prod(t, e1);
		float v = dot(q, dir) / div;
		if (v < 0.0 || v + u > 1.0)
			continue;
		float ts = dot(q, e2) / div;
		if (ts < 0.0)
			continue;
		if (k_min == -1 || ts < ts_min) {
			k_min = k;
			ts_min = ts;
		}
	}
	return { k_min, ts_min };
}


struct int_float_pair {
	int first;
	float second;
};
__device__ int_float_pair find_crossing_gpu(float3 pos, float3 dir, trig* trigs, long long trigsSize , int igTrigIndex = -1, bool ignoreLights = false) {
	int k_min = -1;
	float ts_min;
	for (int k = 0; k < trigsSize; k++) {
		if (k == igTrigIndex) {
			continue;
		}
		trig curTrig = trigs[k];
		if (curTrig.isLight && ignoreLights) {
			continue;
		}
		float3 e1 = curTrig.b - curTrig.a;
		float3 e2 = curTrig.c - curTrig.a;
		float3 p = prod(dir, e2);
		float div = dot(p, e1);
		if (fabs(div) < 1e-10)
			continue;
		float3 t = pos - curTrig.a;
		float u = dot(p, t) / div;
		if (u < 0.0 || u > 1.0)
			continue;
		float3 q = prod(t, e1);
		float v = dot(q, dir) / div;
		if (v < 0.0 || v + u > 1.0)
			continue;
		float ts = dot(q, e2) / div;
		if (ts < 0.0)
			continue;
		if (k_min == -1 || ts < ts_min) {
			k_min = k;
			ts_min = ts;
		}
	}
	return { k_min, ts_min };
}

__host__ uchar4 ray_cpu(float3 pos, float3 dir) {
	auto res = find_crossing(pos, dir);
	int k_min = res.first;
	float ts_min = res.second;

	if (k_min == -1) return { 0, 0, 0, 0 };

	trig& minTrig = TRIGS[k_min];
	
	if (!minTrig.isLight) {
		float3 normal = norm(prod(minTrig.a - minTrig.b, minTrig.a - minTrig.c));
		float3 point = pos + (dir * ts_min);
		float3 result_color{ 0.0f, 0.0f, 0.0f };
		for (light& l : LIGHTS) {
			float3 light_dir = norm(l.position - point);
			auto cross_ans = find_crossing(point, light_dir, k_min, true);
			if (cross_ans.first == -1) {
				float3 ambient = l.color * 0.2f;
				float diffusK = std::max(dot(normal, light_dir), 0.0f);
				float3 diffuse = l.color * minTrig.reflection * diffusK;
				float3 halfwayDir = norm(light_dir + (dir * -1));
				float specK = std::pow(std::max(dot(normal, halfwayDir), 0.0f), 64.0f);
				float3 specular = l.color * 0.7f * specK;
				float3 color;
				color.x = (float)minTrig.color.x / 255.0f;
				color.y = (float)minTrig.color.y / 255.0f;
				color.z = (float)minTrig.color.z / 255.0f;
				float3 result = (ambient + diffuse + specular) * color;
				float distToLight = len(l.position - point);
			if (distToLight > 0.7) {
				float distK = distToLight + 0.3f;
				result = result * (1.0f / (sqrt(distK * distK * distK)));
			}
				result_color = result_color + result * l.intensity;
			}
		}
		if (result_color.x > 1) result_color.x = 1;
		if (result_color.y > 1) result_color.y = 1;
		if (result_color.z > 1) result_color.z = 1;
		if (result_color.x < 0) result_color.x = 0;
		if (result_color.y < 0) result_color.x = 0;
		if (result_color.z < 0) result_color.x = 0;

		uchar x = uchar((float)(result_color.x) * 255.0f);
		uchar y = uchar((float)(result_color.y) * 255.0f);
		uchar z = uchar((float)(result_color.z) * 255.0f);
		return make_uchar4(x, y, z, minTrig.color.w);
	}
	else {
		return make_uchar4(minTrig.color.x, minTrig.color.y, minTrig.color.z, minTrig.color.w);
	}
}

__device__ float device_pow(float b, unsigned long long e) {
	float v = 1.0;
	while (e != 0) {
		if ((e & 1) != 0) {
			v *= b;
		}
		b *= b;
		e >>= 1;
	}
	return v;
}

__device__ uchar4 ray_gpu(float3 pos, float3 dir, trig* trigs, long long trigsSize, light* lights, long long lightsSize) {
	auto res = find_crossing_gpu(pos, dir, trigs, trigsSize);
	int k_min = res.first;
	float ts_min = res.second;

	if (k_min == -1) return { 0, 0, 0, 0 };

	trig minTrig = trigs[k_min];

	if (!minTrig.isLight) {
		float3 normal = norm(prod(minTrig.a - minTrig.b, minTrig.a - minTrig.c));
		float3 point = pos + (dir * ts_min);
		float3 result_color{ 0.0f, 0.0f, 0.0f };
		for (int i = 0; i < lightsSize; ++i) {
			light l = lights[i];
			float3 light_dir = norm(l.position - point);
			auto cross_ans = find_crossing_gpu(point, light_dir, trigs, trigsSize, k_min, true);
			if (cross_ans.first == -1) {
				float3 ambient = l.color * 0.2;
				float diffusK = max(dot(normal, light_dir), 0.0f);
				float3 diffuse = l.color * minTrig.reflection * diffusK;
				float3 halfwayDir = norm(light_dir + (dir * -1));
				float specK = device_pow(max(dot(normal, halfwayDir), 0.0f), 64);
				float3 specular = l.color * 0.7f * specK;
				float3 color;
				color.x = (float)minTrig.color.x / 255.0f;
				color.y = (float)minTrig.color.y / 255.0f;
				color.z = (float)minTrig.color.z / 255.0f;
				float3 result = (ambient + diffuse + specular) * color;
				float distToLight = len(l.position - point);
				if (distToLight > 0.7) {
					float distK = distToLight + 0.3;
					result = result * (1.0f / (sqrt(distK * distK * distK)));
				}
				result_color = result_color + result * l.intensity;
			}
		}
		if (result_color.x > 1) result_color.x = 1;
		if (result_color.y > 1) result_color.y = 1;
		if (result_color.z > 1) result_color.z = 1;
		if (result_color.x < 0) result_color.x = 0;
		if (result_color.y < 0) result_color.x = 0;
		if (result_color.z < 0) result_color.x = 0;

		unsigned char x = (float)(result_color.x) * 255.0f;
		unsigned char y = (float)(result_color.y) * 255.0f;
		unsigned char z = (float)(result_color.z) * 255.0f;
		return make_uchar4(x, y, z, minTrig.color.w);
	}
	else {
		return make_uchar4(minTrig.color.x, minTrig.color.y, minTrig.color.z, minTrig.color.w);
	}
}

inline void rendering_label_cpu(int frame_num, long long pixel_num) {
	LOADING_CASH.print_label_if_percent_significantly_increased(
		(float)pixel_num * 100 / (float)CAMERA_SETTINGS.total_scaled_pixel_count, frame_num, pixel_num
	);
}

void render_cpu(float3 pc, float3 pv, int w, int h, float angle, uchar4* data, int frame_num) {
	int i, j;
	float dw = 2.0f / (w - 1.0f);
	float dh = 2.0f / (h - 1.0f);
	float z = 1.0f / tanf(angle * PI / 360.0f);
	float3 bz = norm(pv - pc);
	float3 bx = norm(prod(bz, { 0.0, 0.0, 1.0 }));
	float3 by = norm(prod(bx, bz));
	for (i = 0; i < w; i++)
		for (j = 0; j < h; j++) {
			int pixel_num = i * h + j;
			rendering_label_cpu(frame_num, pixel_num);
			float3 v = { -1.0f + dw * (float)i, (-1.0f + dh * (float)j) * (float)h / (float)w, (float)z };
			float3 dir = mult(bx, by, bz, v);
			data[(h - 1 - j) * w + i] = ray_cpu(pc, norm(dir));
		}
}

__global__ void render_gpu(
	float3 pc, float3 pv, int w, int h, float angle, uchar4* data,
	trig* trigs, long long trigsSize,
	light* lights, long long lightsSize
) {
	const float PI = 3.141592653589;
	int i, j;
	float dw = 2.0f / (w - 1.0f);
	float dh = 2.0f / (h - 1.0f);
	float z = 1.0f / tanf(angle * PI / 360.0f);
	float3 bz = norm(pv - pc);
	float3 bx = norm(prod(bz, { 0.0, 0.0, 1.0 }));
	float3 by = norm(prod(bx, bz));
	int idX = blockDim.x * blockIdx.x + threadIdx.x;
	int offsetX = blockDim.x * gridDim.x;
	int idY = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetY = blockDim.y * gridDim.y;
	for (i = idX; i < w; i+= offsetX)
		for (j = idY; j < h; j+=offsetY) {
			float3 v = { -1.0f + dw * (float)i, (-1.0f + dh * (float)j) * (float)h / (float)w, (float)z };
			float3 dir = mult(bx, by, bz, v);
			data[(h - 1 - j) * w + i] = ray_gpu(pc, norm(dir), trigs, trigsSize, lights, lightsSize);
		}
}

void cpu_ssaa(uchar4* picture, const uchar4* image, int w, int h, int sqrt_per_pixel) {
	if (sqrt_per_pixel <= 1) {
		memcpy(picture, image, sizeof(uchar4) * w * h);
		return;
	}
	int big_w = w * sqrt_per_pixel;

	for (int i = 0; i < h; ++i) {
		for (int j = 0; j < w; ++j) {
			int start_y = i * sqrt_per_pixel;
			int start_x = j * sqrt_per_pixel;

			longlong3 mean = { 0, 0, 0 };

			// Compute single pixel of picture as average value of window.
			for (int n = start_y; n < start_y + sqrt_per_pixel; ++n) {
				for (int m = start_x; m < start_x + sqrt_per_pixel; ++m) {
					mean.x += image[n * big_w + m].x;
					mean.y += image[n * big_w + m].y;
					mean.z += image[n * big_w + m].z;
				}
			}
			mean.x /= ((long long)sqrt_per_pixel * sqrt_per_pixel);
			mean.y /= ((long long)sqrt_per_pixel * sqrt_per_pixel);
			mean.z /= ((long long)sqrt_per_pixel * sqrt_per_pixel);
			if (mean.x > 255)
				mean.x = 255;
			if (mean.y > 255)
				mean.y = 255;
			if (mean.z > 255)
				mean.z = 255;
			// Write value to picture
			picture[i * w + j] = make_uchar4((uchar)mean.x, (uchar)mean.y, (uchar)mean.z, 255);
		}
	}
}
__global__ void gpu_ssaa(uchar4* picture, const uchar4* image, int w, int h, int sqrt_per_pixel) {
	int big_w = w * sqrt_per_pixel;
	int idX = blockDim.x * blockIdx.x + threadIdx.x;
	int offsetX = blockDim.x * gridDim.x;
	int idY = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetY = blockDim.y * gridDim.y;
	for (int i = idX; i < h; i+=offsetX) {
		for (int j = idY; j < w; j+=offsetY) {
			int start_y = i * sqrt_per_pixel;
			int start_x = j * sqrt_per_pixel;

			longlong3 mean = { 0, 0, 0 };

			// Compute single pixel of picture as average value of window.
			for (int n = start_y; n < start_y + sqrt_per_pixel; ++n) {
				for (int m = start_x; m < start_x + sqrt_per_pixel; ++m) {
					mean.x += image[n * big_w + m].x;
					mean.y += image[n * big_w + m].y;
					mean.z += image[n * big_w + m].z;
				}
			}
			mean.x /= ((long long)sqrt_per_pixel * sqrt_per_pixel);
			mean.y /= ((long long)sqrt_per_pixel * sqrt_per_pixel);
			mean.z /= ((long long)sqrt_per_pixel * sqrt_per_pixel);
			if (mean.x > 255)
				mean.x = 255;
			if (mean.y > 255)
				mean.y = 255;
			if (mean.z > 255)
				mean.z = 255;
			// Write value to picture
			picture[i * w + j] = make_uchar4((uchar)mean.x, (uchar)mean.y, (uchar)mean.z, 255);
		}
	}
}
void cpu_calc() {
	//chrono_timer::timer_sec timer;
	//timer.start();
	std::cout << "CPU CALCULATION\n";
	char buff[256];
	uchar4* data = (uchar4*)malloc(sizeof(uchar4) * CAMERA_SETTINGS.total_scaled_pixel_count);
	uchar4* data_out = (uchar4*)malloc(sizeof(uchar4) * CAMERA_SETTINGS.total_pixel_count);
	for (int frame_number = 0; frame_number < FRAMES_COUNT; frame_number++) {
		float t = SPEED * frame_number + OFFSET;
		float3 pc = CAMERA_MOVEMENT_SETTINGS.positionMovement.vecByT(t);
		float3 pv = CAMERA_MOVEMENT_SETTINGS.viewMovement.vecByT(t);
		render_cpu(
			pc, pv,
			CAMERA_SETTINGS.scaled_width, CAMERA_SETTINGS.scaled_height,
			CAMERA_SETTINGS.angle, data, frame_number
		);

		//ssaa
		cpu_ssaa(data_out, data, CAMERA_SETTINGS.width, CAMERA_SETTINGS.height, SQRT_OF_LIGHTS_COUNT_PER_PIXEL);

		sprintf(buff, OUTPUT_RENDERED_IMAGE_PATH.c_str(), frame_number);
		std::cout << "\r                                                                               ";
		std::cout << "\r" << frame_number + 1<< ": " << buff << "\n";

		FILE* out = fopen(buff, "wb");
		fwrite(&CAMERA_SETTINGS.width, sizeof(int), 1, out);
		fwrite(&CAMERA_SETTINGS.height, sizeof(int), 1, out);
		fwrite(data_out, sizeof(uchar4), (size_t)CAMERA_SETTINGS.width * CAMERA_SETTINGS.height, out);
		fclose(out);
	}
	free(data);
	//std::cout << "\nTIME SPENT : " << timer.time() << "min\n";
}
void gpu_calc() {
	std::cout << "GPU CALCULATION\n";
	//chrono_timer::timer_sec timer;
	//timer.start();
	char buff[256];
	uchar4* data = new uchar4[CAMERA_SETTINGS.total_scaled_pixel_count];
	uchar4* data_out = new uchar4[CAMERA_SETTINGS.total_pixel_count];
	uchar4* data_dev = nullptr, * data_out_dev = nullptr;
	trig* trigs_dev;
	light* lights_dev;

	hipMalloc(&data_dev, sizeof(uchar4) * CAMERA_SETTINGS.total_scaled_pixel_count);

	for (int frame_number = 0; frame_number < FRAMES_COUNT; frame_number++) {
		float t = SPEED * frame_number + OFFSET;
		float3 pc = CAMERA_MOVEMENT_SETTINGS.positionMovement.vecByT(t);
		float3 pv = CAMERA_MOVEMENT_SETTINGS.viewMovement.vecByT(t);
		
		hipMalloc(&trigs_dev, sizeof(trig) * TRIGS.size());
		hipMalloc(&lights_dev, sizeof(light) * LIGHTS.size());
		hipMemcpy(trigs_dev, TRIGS.data(), sizeof(trig) * TRIGS.size(), hipMemcpyHostToDevice);
		hipMemcpy(lights_dev, LIGHTS.data(), sizeof(light) * LIGHTS.size(), hipMemcpyHostToDevice);
		render_gpu <<< BLOCKS, THREADS >>> (
			pc, pv,
			CAMERA_SETTINGS.scaled_width, CAMERA_SETTINGS.scaled_height,
			CAMERA_SETTINGS.angle, data_dev,
			trigs_dev, TRIGS.size(),
			lights_dev, LIGHTS.size()
		);
		auto err = hipGetLastError();
		CEH(err);
		hipFree(lights_dev);
		hipFree(trigs_dev);
		//ssaa
		if (SQRT_OF_LIGHTS_COUNT_PER_PIXEL > 1) {
			hipMalloc(&data_out_dev, sizeof(uchar4) * CAMERA_SETTINGS.total_pixel_count);
			gpu_ssaa << < BLOCKS, THREADS >> > (data_out_dev, data_dev, CAMERA_SETTINGS.width, CAMERA_SETTINGS.height, SQRT_OF_LIGHTS_COUNT_PER_PIXEL);
			hipMemcpy(data_out, data_out_dev, sizeof(uchar4) * CAMERA_SETTINGS.total_pixel_count, hipMemcpyDeviceToHost);
			hipFree(data_out_dev);
		}
		else {
			hipMemcpy(data_out, data_dev, sizeof(uchar4) * CAMERA_SETTINGS.total_pixel_count, hipMemcpyDeviceToHost);
		}
		sprintf(buff, OUTPUT_RENDERED_IMAGE_PATH.c_str(), frame_number);
		std::cout << "\r                                                                               ";
		std::cout << "\r" << "["  << frame_number + 1<< '/' << FRAMES_COUNT <<  "]: " << buff;

		FILE* out = fopen(buff, "wb");
		fwrite(&CAMERA_SETTINGS.width, sizeof(int), 1, out);
		fwrite(&CAMERA_SETTINGS.height, sizeof(int), 1, out);
		fwrite(data_out, sizeof(uchar4), (size_t)CAMERA_SETTINGS.width * CAMERA_SETTINGS.height, out);
		fclose(out);
	}

	hipFree(data_dev);
	delete[] data_out;
	delete[] data;
	//std::cout << "\nTIME SPENT : " << timer.time() << "sec\n";
}


__host__ void add_cube(base_figure figure) {
	std::vector<float3> ver_mas;
	float R = figure.radius;
	ver_mas.push_back({ R / (2 * cosf(30)), R / (2 * cosf(30)), -R / (2 * cosf(30)) });
	ver_mas.push_back({ -R / (2 * cosf(30)), R / (2 * cosf(30)), -R / (2 * cosf(30)) });
	ver_mas.push_back({ -R / (2 * cosf(30)), -R / (2 * cosf(30)), -R / (2 * cosf(30)) });
	ver_mas.push_back({ R / (2 * cosf(30)), -R / (2 * cosf(30)), -R / (2 * cosf(30)) });

	ver_mas.push_back({ R / (2 * cosf(30)), R / (2 * cosf(30)), R / (2 * cosf(30)) });
	ver_mas.push_back({ -R / (2 * cosf(30)), R / (2 * cosf(30)), R / (2 * cosf(30)) });
	ver_mas.push_back({ -R / (2 * cosf(30)), -R / (2 * cosf(30)),  R / (2 * cosf(30)) });
	ver_mas.push_back({ R / (2 * cosf(30)), -R / (2 * cosf(30)),  R / (2 * cosf(30)) });


	for (auto& el : ver_mas) {
		el.x += figure.position.x;
		el.y += figure.position.y;
		el.z += figure.position.z;
	}

	uchar4 color = { 
		uchar(figure.color.x * 255.0f),
		uchar(figure.color.y * 255.0f),
		uchar(figure.color.z * 255.0f),
		255
	};

	bool isLight = figure.isLight;
	TRIGS.push_back(trig{ ver_mas[1], ver_mas[4], ver_mas[0], color, isLight, figure.reflection });//-
	TRIGS.push_back(trig{ ver_mas[5], ver_mas[4], ver_mas[1], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[1], ver_mas[2], ver_mas[5], color, isLight, figure.reflection });//+
	TRIGS.push_back(trig{ ver_mas[2], ver_mas[6], ver_mas[5], color, isLight, figure.reflection });//+

	TRIGS.push_back(trig{ ver_mas[3], ver_mas[6], ver_mas[2], color, isLight, figure.reflection });//-
	TRIGS.push_back(trig{ ver_mas[7], ver_mas[6], ver_mas[3], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[7], ver_mas[3], ver_mas[0], color, isLight, figure.reflection });//+
	TRIGS.push_back(trig{ ver_mas[4], ver_mas[7], ver_mas[0], color, isLight, figure.reflection });//+

	TRIGS.push_back(trig{ ver_mas[4], ver_mas[5], ver_mas[6], color, isLight, figure.reflection });//+
	TRIGS.push_back(trig{ ver_mas[6], ver_mas[7], ver_mas[4], color, isLight, figure.reflection });//+

	TRIGS.push_back(trig{ ver_mas[0], ver_mas[2], ver_mas[1], color, isLight, figure.reflection });//+
	TRIGS.push_back(trig{ ver_mas[0], ver_mas[3], ver_mas[2], color, isLight, figure.reflection });//+
}
__host__ void add_icosahedron(base_figure figure) // определить массив вершин
{
	// начальные значения
	float R = figure.radius;
	float a = 4 * R / sqrtf(10 + 2 * sqrtf(5)); // сторона икосаэдра
	float alpha = acosf((1 - a * a / 2 / R / R)); // первый угол поворота по тэта
	float k = PI / 180.0f;
	// точки икосаэдра
	std::vector<float3> ver_mas;
	ver_mas.push_back({ 0,0, R });
	ver_mas.push_back(float3{ R * sinf(alpha) * sinf(0), R * sinf(alpha) * cosf(0), R * cosf(alpha) });
	ver_mas.push_back({ R * sinf(alpha) * sinf(72 * k), R * sinf(alpha) * cosf(72 * k), R * cosf(alpha) });
	ver_mas.push_back({ R * sinf(alpha) * sinf(2 * 72 * k), R * sinf(alpha) * cosf(2 * 72 * k), R * cosf(alpha) });
	ver_mas.push_back({ R * sinf(alpha) * sinf(3 * 72 * k), R * sinf(alpha) * cosf(3 * 72 * k), R * cosf(alpha) });
	ver_mas.push_back({ R * sinf(alpha) * sinf(4 * 72 * k), R * sinf(alpha) * cosf(4 * 72 * k), R * cosf(alpha) });
	ver_mas.push_back({ R * sinf(PI - alpha) * sinf(-36 * k), R * sinf(PI - alpha) * cosf(-36 * k), R * cosf(PI - alpha) });
	ver_mas.push_back({ R * sinf(PI - alpha) * sinf(36 * k), R * sinf(PI - alpha) * cosf(36 * k),  R * cosf(PI - alpha) });
	ver_mas.push_back({ R * sinf(PI - alpha) * sinf((36 + 72) * k), R * sinf(PI - alpha) * cosf((36 + 72) * k), R * cosf(PI - alpha) });
	ver_mas.push_back({ R * sinf(PI - alpha) * sinf((36 + 2 * 72) * k), R * sinf(PI - alpha) * cosf((36 + 2 * 72) * k), R * cosf(PI - alpha) });
	ver_mas.push_back({ R * sinf(PI - alpha) * sinf((36 + 3 * 72) * k), R * sinf(PI - alpha) * cosf((36 + 3 * 72) * k), R * cosf(PI - alpha) });
	ver_mas.push_back({ 0, 0, -R });

	for (auto& el : ver_mas) {
		el.x += figure.position.x;
		el.y += figure.position.y;
		el.z += figure.position.z;
	}

	bool isLight = figure.isLight;
	uchar4 color = { 
		uchar(figure.color.x * 255.0f),
		uchar(figure.color.y * 255.0f), 
		uchar(figure.color.z * 255.0f),
		255
	};
	TRIGS.push_back(trig{ ver_mas[0], ver_mas[2], ver_mas[1], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[0], ver_mas[3], ver_mas[2], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[0], ver_mas[4], ver_mas[3], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[0], ver_mas[5], ver_mas[4], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[0], ver_mas[1], ver_mas[5], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[6], ver_mas[1], ver_mas[7], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[7], ver_mas[1], ver_mas[2], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[7], ver_mas[2], ver_mas[8], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[8], ver_mas[2], ver_mas[3], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[8], ver_mas[3], ver_mas[9], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[9], ver_mas[3], ver_mas[4], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[9], ver_mas[4], ver_mas[10], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[10], ver_mas[4], ver_mas[5], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[10], ver_mas[5], ver_mas[6], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[6], ver_mas[5], ver_mas[1], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[7], ver_mas[11], ver_mas[6], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[8], ver_mas[11], ver_mas[7], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[9], ver_mas[11], ver_mas[8], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[10], ver_mas[11], ver_mas[9], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[6], ver_mas[11], ver_mas[10], color, isLight, figure.reflection });
}
__host__ void add_octahedron(base_figure figure) {
	std::vector<float3> ver_mas;
	float R = figure.radius;
	ver_mas.push_back({ 0, 0, R });
	ver_mas.push_back({ 0, 0, -R });
	ver_mas.push_back({ R * sinf(45), R * cosf(45), 0 });
	ver_mas.push_back({ -R * sinf(45), R * cosf(45), 0 });
	ver_mas.push_back({ -R * sinf(45), -R * cosf(45), 0 });
	ver_mas.push_back({ R * sinf(45), -R * cosf(45), 0 });

	for (auto& el : ver_mas) {
		el.x += figure.position.x;
		el.y += figure.position.y;
		el.z += figure.position.z;
	}

	uchar4 color = { 
		uchar(figure.color.x * 255.0f),
		uchar(figure.color.y * 255.0f),
		uchar(figure.color.z * 255.0f),
		255
	};

	bool isLight = figure.isLight;
	TRIGS.push_back(trig{ ver_mas[0], ver_mas[3], ver_mas[2], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[0], ver_mas[4], ver_mas[3], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[0], ver_mas[5], ver_mas[4], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[0], ver_mas[2], ver_mas[5], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[1], ver_mas[2], ver_mas[3], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[1], ver_mas[3], ver_mas[4], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[1], ver_mas[4], ver_mas[5], color, isLight, figure.reflection });
	TRIGS.push_back(trig{ ver_mas[1], ver_mas[5], ver_mas[2], color, isLight, figure.reflection });
}
__host__ void add_floor() {
	uchar4 color = { 
		uchar(FLOOR.color.x * 255.0f),
		uchar(FLOOR.color.y * 255.0f),
		uchar(FLOOR.color.z * 255.0f),
		255 
	};
	TRIGS.push_back(trig{ FLOOR.point1, FLOOR.point3, FLOOR.point2, color, false, FLOOR.reflection });
	TRIGS.push_back(trig{ FLOOR.point1, FLOOR.point4, FLOOR.point3, color, false, FLOOR.reflection });
}

void build_space() {
	add_cube(FIGURE_1);
	add_octahedron(FIGURE_2);
	add_icosahedron(FIGURE_3);
	for (int i = 0; i < LIGHTS_COUNT; ++i) {
		add_cube(base_figure::fromLight(LIGHTS[i]));
	}
	add_floor();
}

int main(int argc, char* argv[]) {
	std::ios::sync_with_stdio(false);
	key_config key_cfg(argc, argv);

	input();
	init();
	build_space();

	if (key_cfg.is_default) {
		//возвращаем наиболее красивый результат
		default_calc();
	}
	else if (key_cfg.is_cpu) {
		//считаем на cpu
		cpu_calc();
	}
	else {
		//считаем на gpu
		gpu_calc();
	}
	//system("python big_conv.py \\out");
	//system("C:\\dev\\CUDA\\KP\\KP\\1\\ffmpeg-master-latest-win64-gpl-shared\\bin\\ffmpeg.exe -y -i C:\\dev\\CUDA\\KP\\KP\\1\\out\\img_%d.jpg -loop 0 -filter:v setpts=2*PTS C:\\dev\\CUDA\\KP\\KP\\1\\out\\res.gif");
	exit(0);
}
