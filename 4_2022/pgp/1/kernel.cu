#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

#define CUDA_ERROR_HANDLER(err) do { if (err != hipSuccess) { std::cout << hipGetErrorString(err) << std::endl; exit(0);} } while (0)
#define CEH(err) CUDA_ERROR_HANDLER(err)

#define BLOCKS dim3(100,100)
#define THREADS 1024
#define min(a, b) a < b ? a : b

__global__ void minKernel(long long n, const double* a, const double* b, double* c) {
    long long i = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;
    for (; i < n; i += offset) {
        c[i] = min(a[i], b[i]);
    }
}

int main() {
    long long n;
    std::cin >> n;
    double* a, * b, *c;
    double* dev_a, * dev_b, * dev_c = 0;

    a = new double[n];
    b = new double[n];
    c = new double[n];

    for (long long i = 0; i < n; ++i) {
        std::cin >> a[i];
    }

    for (long long i = 0; i < n; ++i) {
        std::cin >> b[i];
    }

    CUDA_ERROR_HANDLER(hipMalloc((void**)&dev_a, n * sizeof(double)));
    CUDA_ERROR_HANDLER(hipMalloc((void**)&dev_b, n * sizeof(double)));
    CUDA_ERROR_HANDLER(hipMalloc((void**)&dev_c, n * sizeof(double)));

    CUDA_ERROR_HANDLER(hipMemcpy(dev_a, a, n * sizeof(double), hipMemcpyHostToDevice));
    CUDA_ERROR_HANDLER(hipMemcpy(dev_b, b, n * sizeof(double), hipMemcpyHostToDevice));

    minKernel <<<BLOCKS, THREADS>>> (n , dev_a, dev_b, dev_c); 
    CUDA_ERROR_HANDLER(hipDeviceSynchronize());

    CUDA_ERROR_HANDLER(hipMemcpy(c, dev_c, n * sizeof(double), hipMemcpyDeviceToHost));
    
    std::cout << std::scientific;
    std::cout.precision(11);
    for (long long i = 0; i < n; ++i) {
        std::cout << c[i] << ' ';
    }

    CUDA_ERROR_HANDLER(hipFree(dev_c));
    CUDA_ERROR_HANDLER(hipFree(dev_a));
    CUDA_ERROR_HANDLER(hipFree(dev_b));
    delete a;
    delete b;
    delete c;
    
    exit(0);
}
