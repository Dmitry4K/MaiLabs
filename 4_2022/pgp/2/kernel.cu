#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<texture_types.h>
#include<>
#include<iostream>
#include<fstream>
#include<string>
#pragma once
#include<chrono>
template<class T = std::chrono::nanoseconds>
class Timer {
    std::chrono::steady_clock::time_point start_point = std::chrono::high_resolution_clock::now();
    std::chrono::steady_clock::time_point end_point = start_point;
    bool is_stoped = true;
public:
    using time_type = T;
    void Start() {
        start_point = std::chrono::high_resolution_clock::now();
        is_stoped = false;
    }
    void Stop() {
        end_point = std::chrono::high_resolution_clock::now();
        is_stoped = true;
    }
    long long Time() {
        return is_stoped ? std::chrono::duration_cast<time_type>(end_point - start_point).count() : std::chrono::duration_cast<time_type>(std::chrono::high_resolution_clock::now() - start_point).count();
    }
};

#define CUDA_ERROR_HANDLER(err) do { if (err != hipSuccess) { std::cout << hipGetErrorString(err) << std::endl; exit(0);} } while (0)
#define CEH(err) CUDA_ERROR_HANDLER(err)

#define BLOCKS dim3(4, 4)
#define THREADS dim3(4, 4)

#define TO_BLACK_WHITE(p) double((0.299*(double)p.x) + (0.587 * (double) p.y) + (0.114* (double)p.z))

texture<uchar4, hipTextureType2D, hipReadModeElementType> tex;

inline uchar4 getByIndexes(uchar4 * image, int x, int y, int w) {
    return image[y * w + x];
}

void previttaLinear(unsigned long long width, unsigned long long height, uchar4 * out, uchar4* image) {
    for (long long y = 1; y < height-1; y++) {
        for (long long x = 1; x < width-1; x++) {
            double gx = 0.0
                + TO_BLACK_WHITE(getByIndexes(image, x + 1, y - 1, width))
                + TO_BLACK_WHITE(getByIndexes(image, x + 1, y, width))
                + TO_BLACK_WHITE(getByIndexes(image, x + 1, y + 1, width))
                - TO_BLACK_WHITE(getByIndexes(image, x - 1, y - 1, width))
                - TO_BLACK_WHITE(getByIndexes(image, x - 1, y, width))
                - TO_BLACK_WHITE(getByIndexes(image, x - 1, y + 1, width));
            double gy = 0.0
                + TO_BLACK_WHITE(getByIndexes(image, x - 1, y + 1, width))
                + TO_BLACK_WHITE(getByIndexes(image, x, y + 1, width))
                + TO_BLACK_WHITE(getByIndexes(image, x + 1, y + 1, width))
                - TO_BLACK_WHITE(getByIndexes(image, x - 1, y - 1, width))
                - TO_BLACK_WHITE(getByIndexes(image, x, y - 1, width))
                - TO_BLACK_WHITE(getByIndexes(image, x + 1, y - 1, width));

            float r = (sqrt(gx * gx + gy * gy));
            if (r > 255.0) {
                r = 255;
            }
            out[y * width + x] = make_uchar4(r, r, r, getByIndexes(image, x, y, width).w);
        }
    }
}
__global__ void previttaKernel(unsigned long long width, unsigned long long height, uchar4* out) {
	long long idx = blockDim.x * blockIdx.x + threadIdx.x;
    long long idy = blockDim.y * blockIdx.y + threadIdx.y;
    long long offsetx = blockDim.x * gridDim.x;
    long long offsety = blockDim.y * gridDim.y;
    for (long long y = idy; y < height; y += offsety) {
        for (long long x = idx; x < width; x += offsetx) {
            double gx = 0.0
                    + TO_BLACK_WHITE(tex2D(tex, x + 1, y - 1))
                    + TO_BLACK_WHITE(tex2D(tex, x + 1, y    ))
                    + TO_BLACK_WHITE(tex2D(tex, x + 1, y + 1))
                    - TO_BLACK_WHITE(tex2D(tex, x - 1, y - 1))
                    - TO_BLACK_WHITE(tex2D(tex, x - 1, y    ))
                    - TO_BLACK_WHITE(tex2D(tex, x - 1, y + 1));
            double gy = 0.0
                    + TO_BLACK_WHITE(tex2D(tex, x - 1, y + 1))
                    + TO_BLACK_WHITE(tex2D(tex, x    , y + 1))
                    + TO_BLACK_WHITE(tex2D(tex, x + 1, y + 1))
                    - TO_BLACK_WHITE(tex2D(tex, x - 1, y - 1))
                    - TO_BLACK_WHITE(tex2D(tex, x    , y - 1))
                    - TO_BLACK_WHITE(tex2D(tex, x + 1, y - 1));

            float r = (sqrt(gx*gx + gy*gy));
            if (r > 255.0) {
                r = 255;
            }
			out[y * width + x] = make_uchar4(r, r, r, tex2D(tex, x, y).w);
		}
	}
}

int main() {
	std::string input_file_path, output_file_path;
	std::cin >> input_file_path >> output_file_path;
	std::ifstream input{ input_file_path , std::ios::in | std::ios::binary};
	unsigned int w, h;
    uchar4* image = nullptr;
    Timer<std::chrono::microseconds> timer;

    //reading
	if (input.is_open()) {
		input.read((char*)&w, sizeof(int));
		input.read((char*)&h, sizeof(int));
		image = new uchar4[(size_t)w * (size_t)h];
		input.read((char*)image, sizeof(uchar4) * (size_t)w * (size_t)h);
        input.close();
    }
	else {
		exit(1);
	}
	//malloc
    hipArray* arr;
    uchar4* out;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	CEH(hipMallocArray(&arr, &ch, w, h));

	//coping to device
	CEH(hipMemcpyToArray(arr, 0, 0, image, sizeof(uchar4) * (size_t)w * (size_t)h, hipMemcpyHostToDevice));

	//config
    tex.addressMode[0] = hipAddressModeClamp;	// Политика обработки выхода за границы по каждому измерению
    tex.addressMode[1] = hipAddressModeClamp;
    tex.channelDesc = ch;
    tex.filterMode = hipFilterModePoint;		// Без интерполяции при обращении по дробным координатам
    tex.normalized = false;

    //binding
    CEH(hipBindTextureToArray(tex, arr, ch));
    CEH(hipMalloc(&out, sizeof(uchar4) * (size_t)w * (size_t)h));

    //executing
    timer.Start();
    previttaKernel <<< BLOCKS, THREADS>>> (w, h, out);
    CEH(hipGetLastError());
    std::cout << "CUDA time: "<< timer.Time() << std::endl;
    uchar4* outl = new uchar4[w * h];
    timer.Start();
    previttaLinear(w, h, outl, image);
    std::cout << "Liner time: " << timer.Time();
    CEH(hipMemcpy(image, out, sizeof(uchar4) * (size_t)w * (size_t)h, hipMemcpyDeviceToHost));

	//formatting result
	std::ofstream output{ output_file_path ,std::ios::out | std::ios::binary};


    output.write((char*)&w, sizeof(int));
	output.write((char*)&h, sizeof(int));
    output.write((char*)image, sizeof(uchar4) * (size_t)w * (size_t)h );
    output.close();
    //unbinding
    CEH(hipUnbindTexture(tex));
    CEH(hipFreeArray(arr));
    CEH(hipFree(out));

    delete[] image;
	exit(0);
}
