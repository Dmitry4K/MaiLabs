#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <mpi.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>
#include <stdio.h>
#include <string>
#include <sstream>

//#define IOSTREAM

#define BLOCKS dim3(8,8,8)
#define THREADS dim3(8,8,8)
#define BLOCKS_1D 256
#define THREADS_1D 256
#define CUDA_ERROR_HANDLER(err) do { if (err != hipSuccess) { std::cout << hipGetErrorString(err) << std::endl; exit(1);} } while (0)
#define CEH(err) CUDA_ERROR_HANDLER(err)

namespace SIDES {
    int LEFT = 0;
    int RIGHT = 1;
    int UP = 2;
    int DOWN = 3;
    int FRONT = 4;
    int BACK = 5;
}

struct vec3 { int x, y, z; };

int blockId, blocksTotalCount, procNameLen, stringLen;

std::string outFileName = "";

double epsilon, lX, lY, lZ, u0, uLeft, uRight, uUp, uBack, uFront, uDown;

vec3 inBlokcPos, blockPos, gridEqDim, blockEqDim;

#define _idx(i, j, k) ( ((k) + 1) * (blockEqDim.y + 2) * (blockEqDim.x + 2) + ((j) + 1) * (blockEqDim.x + 2) + (i) + 1 )
#define _idxblock(i, j, k) ( (k) * (gridEqDim.x * gridEqDim.y) + (j) * gridEqDim.x + (i) )
#define _idblockx(id) (( (id) % (gridEqDim.x * gridEqDim.y) ) % gridEqDim.x)
#define _idblocky(id) (( (id) % (gridEqDim.x * gridEqDim.y) ) / gridEqDim.x)
#define _idblockz(id) (  (id) / (gridEqDim.x * gridEqDim.y) )

#define _idx_kernel(i,j,k,lX,lY,lZ) ( ((k) + 1) * (lY + 2) * (lX + 2) + ((j) + 1) * (lX + 2) + (i) + 1 )

void printToCerr(const char* str) {
    fprintf(stderr, "[%d]%s\n", blockId, str);
}

vec3 getPositionByBlockId(int blockId) {
    int z = blockId / (gridEqDim.x * gridEqDim.y);
    int y = (blockId - z * (gridEqDim.x * gridEqDim.y)) / (gridEqDim.x);
    int x = blockId - z * (gridEqDim.x * gridEqDim.y) - y * gridEqDim.x;
    //fprintf(stderr, "[%d]%d,%d,%d\n", blockId, x, y, z);
    return { x, y, z };
}

int getTagForSideByBlockId(int side, int blockId) {
    return blockId * 10 + side;
}

int getTagForPrinting(int blockId, int indexZY) {
    return blockId * blockEqDim.z * blockEqDim.y + indexZY;
}

bool isLeftSideBlock(int blockId) { return getPositionByBlockId(blockId).x == 0; }
bool isRightSideBlock(int blockId) { return getPositionByBlockId(blockId).x == gridEqDim.x - 1; }
bool isFrontSideBlock(int blockId) { return getPositionByBlockId(blockId).y == 0; }
bool isBackSideBlock(int blockId) { return getPositionByBlockId(blockId).y == gridEqDim.y - 1; }
bool isUpSideBlock(int blockId) { return getPositionByBlockId(blockId).z == gridEqDim.z - 1; }
bool isDownSideBlock(int blockId) { return getPositionByBlockId(blockId).z == 0; }

void sendSide(double* buff, long long buffCount, int destBlockX, int destBlockY, int destBlockZ, int side) {
    MPI_Request request;
    MPI_Status status;
    //fprintf(stderr, "[%d]s%d,%d,%d\n", blockId, destBlockX, destBlockY, destBlockZ);
    int destBlockId = _idxblock(destBlockX, destBlockY, destBlockZ);
    MPI_Isend(buff, buffCount, MPI_DOUBLE, destBlockId, getTagForSideByBlockId(side, blockId), MPI_COMM_WORLD, &request);
    MPI_Wait(&request, &status);
}

void recvSide(double* buff, long long buffCount, int destBlockX, int destBlockY, int destBlockZ, int side) {
    MPI_Request request;
    MPI_Status status;
    //fprintf(stderr, "[%d]r%d,%d,%d\n", blockId, destBlockX, destBlockY, destBlockZ);
    int destBlockId = _idxblock(destBlockX, destBlockY, destBlockZ);
    MPI_Irecv(buff, buffCount, MPI_DOUBLE, destBlockId, getTagForSideByBlockId(side, destBlockId), MPI_COMM_WORLD, &request);
    MPI_Wait(&request, &status);
}

void reciveAndSendLeftSide(double* block) {
    //printToCerr("left exchanging");
    long long buffCount = (long long)(blockEqDim.z + 2) * (blockEqDim.y + 2);
    double* buff = new double[buffCount];
    if (!isLeftSideBlock(blockId)) {
        for (int k = -1; k <= blockEqDim.z; ++k) {
            for (int j = -1; j <= blockEqDim.y; ++j) {
                buff[(k + 1) * (blockEqDim.y + 2) + j + 1] = block[_idx(0, j, k)];
            }
        }
        sendSide(buff, buffCount, blockPos.x - 1, blockPos.y, blockPos.z, SIDES::LEFT);
    }
    if (!isRightSideBlock(blockId)) {
        recvSide(buff, buffCount, blockPos.x + 1, blockPos.y, blockPos.z, SIDES::LEFT);
        for (int k = -1; k <= blockEqDim.z; ++k) {
            for (int j = -1; j <= blockEqDim.y; ++j) {
                block[_idx(blockEqDim.x, j, k)] = buff[(k + 1) * (blockEqDim.y + 2) + j + 1];
            }
        }
    }
    else {
        //fill right side uRight
        for (int k = -1; k <= blockEqDim.z; ++k) {
            for (int j = -1; j <= blockEqDim.y; ++j) {
                block[_idx(blockEqDim.x, j, k)] = uRight;
            }
        }
    }
    delete[] buff;
}
void reciveAndSendRightSide(double* block) {
    //printToCerr("right exchanging");
    long long buffCount = (long long)(blockEqDim.z + 2) * (blockEqDim.y + 2);
    double* buff = new double[buffCount];
    if (!isRightSideBlock(blockId)) {
        for (int k = -1; k <= blockEqDim.z; ++k) {
            for (int j = -1; j <= blockEqDim.y; ++j) {
                buff[(k + 1) * (blockEqDim.y + 2) + j + 1] = block[_idx(blockEqDim.x - 1, j, k)];
            }
        }
        sendSide(buff, buffCount, blockPos.x + 1, blockPos.y, blockPos.z, SIDES::RIGHT);
    }
    if (!isLeftSideBlock(blockId)) {
        recvSide(buff, buffCount, blockPos.x - 1, blockPos.y, blockPos.z, SIDES::RIGHT);
        for (int k = -1; k <= blockEqDim.z; ++k) {
            for (int j = -1; j <= blockEqDim.y; ++j) {
                block[_idx(-1, j, k)] = buff[(k + 1) * (blockEqDim.y + 2) + j + 1];
            }
        }
    }
    else {
        //fill left side
        for (int k = -1; k <= blockEqDim.z; ++k) {
            for (int j = -1; j <= blockEqDim.y; ++j) {
                block[_idx(-1, j, k)] = uLeft;
            }
        }
    }
    delete[] buff;
}
void reciveAndSendUpSide(double* block) {
    //printToCerr("up exchanging");
    long long buffCount = (long long)(blockEqDim.y + 2) * (blockEqDim.x + 2);
    double* buff = new double[buffCount];
    if (!isUpSideBlock(blockId)) {
        for (int j = -1; j <= blockEqDim.y; ++j) {
            for (int i = -1; i <= blockEqDim.x; ++i) {
                buff[(j + 1) * (blockEqDim.x + 2) + i + 1] = block[_idx(i, j, blockEqDim.z - 1)];
            }
        }
        sendSide(buff, buffCount, blockPos.x, blockPos.y, blockPos.z + 1, SIDES::UP);
    }
    if (!isDownSideBlock(blockId)) {
        recvSide(buff, buffCount, blockPos.x, blockPos.y, blockPos.z - 1, SIDES::UP);
        for (int j = -1; j <= blockEqDim.y; ++j) {
            for (int i = -1; i <= blockEqDim.x; ++i) {
                block[_idx(i, j, -1)] = buff[(j + 1) * (blockEqDim.x + 2) + i + 1];
            }
        }
    }
    else {
        // fill Down side
        for (int j = -1; j <= blockEqDim.y; ++j) {
            for (int i = -1; i <= blockEqDim.x; ++i) {
                block[_idx(i, j, -1)] = uDown;
            }
        }
    }
    delete[] buff;
}
void reciveAndSendDownSide(double* block) {
    //printToCerr("down exchanging");
    long long buffCount = (long long)(blockEqDim.y + 2) * (blockEqDim.x + 2);
    double* buff = new double[buffCount];
    if (!isDownSideBlock(blockId)) {
        for (int j = -1; j <= blockEqDim.y; ++j) {
            for (int i = -1; i <= blockEqDim.x; ++i) {
                buff[(j + 1) * (blockEqDim.x + 2) + i + 1] = block[_idx(i, j, 0)];
            }
        }
        sendSide(buff, buffCount, blockPos.x, blockPos.y, blockPos.z - 1, SIDES::DOWN);
    }
    if (!isUpSideBlock(blockId)) {
        recvSide(buff, buffCount, blockPos.x, blockPos.y, blockPos.z + 1, SIDES::DOWN);
        for (int j = -1; j <= blockEqDim.y; ++j) {
            for (int i = -1; i <= blockEqDim.x; ++i) {
                block[_idx(i, j, blockEqDim.z)] = buff[(j + 1) * (blockEqDim.x + 2) + i + 1];
            }
        }
    }
    else {
        // fill Up side
        for (int j = -1; j <= blockEqDim.y; ++j) {
            for (int i = -1; i <= blockEqDim.x; ++i) {
                block[_idx(i, j, blockEqDim.z)] = uUp;
            }
        }
    }
    delete[] buff;
}
void reciveAndSendFrontSide(double* block) {
    //printToCerr("front exchanging");
    long long buffCount = (long long)(blockEqDim.z + 2) * (blockEqDim.x + 2);
    double* buff = new double[buffCount];
    if (!isFrontSideBlock(blockId)) {
        for (int k = -1; k <= blockEqDim.z; ++k) {
            for (int i = -1; i <= blockEqDim.x; ++i) {
                buff[(k + 1) * (blockEqDim.x + 2) + i + 1] = block[_idx(i, 0, k)];
            }
        }
        sendSide(buff, buffCount, blockPos.x, blockPos.y - 1, blockPos.z, SIDES::FRONT);
    }
    if (!isBackSideBlock(blockId)) {
        recvSide(buff, buffCount, blockPos.x, blockPos.y + 1, blockPos.z, SIDES::FRONT);
        for (int k = -1; k <= blockEqDim.z; ++k) {
            for (int i = -1; i <= blockEqDim.x; ++i) {
                block[_idx(i, blockEqDim.y, k)] = buff[(k + 1) * (blockEqDim.x + 2) + i + 1];
            }
        }
    }
    else {
        // fill back side
        for (int k = -1; k <= blockEqDim.z; ++k) {
            for (int i = -1; i <= blockEqDim.x; ++i) {
                block[_idx(i, blockEqDim.y, k)] = uBack;
            }
        }
    }
    delete[] buff;
}
void reciveAndSendBackSide(double* block) {
    //printToCerr("back exchanging");
    long long buffCount = (long long)(blockEqDim.z + 2) * (blockEqDim.x + 2);
    double* buff = new double[buffCount];
    if (!isBackSideBlock(blockId)) {
        for (int k = -1; k <= blockEqDim.z; ++k) {
            for (int i = -1; i <= blockEqDim.x; ++i) {
                buff[(k + 1) * (blockEqDim.x + 2) + i + 1] = block[_idx(i, blockEqDim.y - 1, k)];
            }
        }
        sendSide(buff, buffCount, blockPos.x, blockPos.y + 1, blockPos.z, SIDES::BACK);
    }
    if (!isFrontSideBlock(blockId)) {
        recvSide(buff, buffCount, blockPos.x, blockPos.y - 1, blockPos.z, SIDES::BACK);
        for (int k = -1; k <= blockEqDim.z; ++k) {
            for (int i = -1; i <= blockEqDim.x; ++i) {
                block[_idx(i, -1, k)] = buff[(k + 1) * (blockEqDim.x + 2) + (i + 1)];
            }
        }
    }
    else {
        // fill front side
        for (int k = -1; k <= blockEqDim.z; ++k) {
            for (int i = -1; i <= blockEqDim.x; ++i) {
                block[_idx(i, -1, k)] = uFront;
            }
        }
    }
    delete[] buff;
}

void printExclusiveBlock(double* block) {
    for (int k = -1; k <= blockEqDim.z; k++) {
        for (int j = -1; j <= blockEqDim.y; j++) {
            for (int i = -1; i <= blockEqDim.x; i++) {
                std::cout << block[_idx(i, j, k)] << ' ';
            }
            std::cout << '|';
        }
        std::cout << '\n';
    }
}

void printBlock(double* block) {
    for (int k = 0; k < blockEqDim.z; k++) {
        for (int j = 0; j < blockEqDim.y; j++) {
            for (int i = 0; i < blockEqDim.x; i++) {
                std::cout << block[_idx(i, j, k)] << ' ';
            }
            std::cout << '|';
        }
        std::cout << '\n';
    }
}
void printGlobalVariableToCerr() {
    if (blockId == 0) {
        //std::cerr << "Global vars\n";
        std::cerr << gridEqDim.x << ' ' << gridEqDim.y << ' ' << gridEqDim.z << ' ';
        std::cerr << blockEqDim.x << ' ' << blockEqDim.y << ' ' << blockEqDim.z << ' ';
        std::cerr << outFileName << ' ' << epsilon << ' ';
        std::cerr << lX << ' ' << lY << ' ' << lZ << ' ';
        std::cerr << uDown << ' ' << uUp << ' ' << uLeft << ' ' << uRight << ' ' << uFront << ' ' << uBack << ' ';
        std::cerr << u0 << std::endl;
    }
}

void multiPrint(double* data) {
    //int sizeOfSingleElementIs = 13;
    std::stringstream stringStream;
    stringStream << std::scientific << std::setprecision(6);

    int bufferSize = 20;
    for (int k = 0; k < blockEqDim.z; k++) {
        for (int j = 0; j < blockEqDim.y; j++) {
            for (int i = 0; i < blockEqDim.x; i++) {
                int prev = stringStream.str().length();
                stringStream << data[_idx(i, j, k)];
                int after = stringStream.str().length();
                for (int n = 0; n < bufferSize - (after - prev)-1; ++n) {
                    stringStream << ' ';
                }
                if (i == blockEqDim.x - 1 && blockPos.x == gridEqDim.x - 1) {
                    stringStream << '\n';
                }
                else {
                    stringStream << ' ';
                }
            }
        }
    }
    MPI_Datatype filetype;
    int array_of_sizes[3] = {
        blockEqDim.z * gridEqDim.z, blockEqDim.y * gridEqDim.y, blockEqDim.x * gridEqDim.x * bufferSize
    };
    int array_of_subsizes[3] = { blockEqDim.z, blockEqDim.y, blockEqDim.x * bufferSize };
    int array_of_starts[3] = {
        blockPos.z * blockEqDim.z, blockPos.y * blockEqDim.y, blockPos.x * blockEqDim.x * bufferSize
    };
    MPI_Type_create_subarray(3, array_of_sizes, array_of_subsizes, array_of_starts, MPI_ORDER_C, MPI_CHAR, &filetype);
    MPI_Type_commit(&filetype);  

    MPI_File fp;
    MPI_File_delete(outFileName.c_str(), MPI_INFO_NULL);
    MPI_File_open(MPI_COMM_WORLD, outFileName.c_str(), MPI_MODE_CREATE | MPI_MODE_WRONLY, MPI_INFO_NULL, &fp);
    MPI_File_set_view(fp, 0, MPI_CHAR, filetype, "native", MPI_INFO_NULL);
    MPI_File_write_all(fp, stringStream.str().c_str(), stringStream.str().length(), MPI_CHAR, MPI_STATUS_IGNORE);
    MPI_File_close(&fp);
}

void castFileName() {
    char tmpChar= ' ';
    MPI_Bcast(&stringLen, 1, MPI_INT, 0, MPI_COMM_WORLD);
    if (blockId != 0) {
        outFileName.resize(stringLen);
    }
    for (int i = 0; i < stringLen; ++i) {
        if (blockId == 0) {
            tmpChar = outFileName[i];
        }
        MPI_Bcast(&tmpChar, 1, MPI_CHAR, 0, MPI_COMM_WORLD);
        outFileName[i] = tmpChar;
    }
}

__global__ void matrix_calc_kernel(double* uCurr, double* uNext, double* diffs, int lenX, int lenY, int lenZ, double hX, double hY, double hZ) {

    int idX = blockDim.x * blockIdx.x + threadIdx.x;
    int offsetX = blockDim.x * gridDim.x;
    int idY = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetY = blockDim.y * gridDim.y;
    int idZ = blockDim.z * blockIdx.z + threadIdx.z;
    int offsetZ = blockDim.z * gridDim.z;
    int i, j, k;
    for (i = idX; i < lenX; i+= offsetX) {
        for (j = idY; j < lenY; j+= offsetY) {
            for (k = idZ; k < lenZ; k+= offsetZ) {
                uNext[_idx_kernel(i, j, k, lenX, lenY, lenZ)] =
                    ((uCurr[_idx_kernel(i + 1, j, k, lenX, lenY, lenZ)] + uCurr[_idx_kernel(i - 1, j, k, lenX, lenY, lenZ)]) / (hX * hX) +
                        (uCurr[_idx_kernel(i, j + 1, k, lenX, lenY, lenZ)] + uCurr[_idx_kernel(i, j - 1, k, lenX, lenY, lenZ)]) / (hY * hY) +
                        (uCurr[_idx_kernel(i, j, k + 1, lenX, lenY, lenZ)] + uCurr[_idx_kernel(i, j, k - 1, lenX, lenY, lenZ)]) / (hZ * hZ)) /
                    (2.0 * ((1.0 / (hX * hX)) + (1.0 / (hY * hY)) + (1.0 / (hZ * hZ))));
                diffs[k * lenY * lenX + j * lenX + i] = abs(
                    uNext[_idx_kernel(i, j, k, lenX, lenY, lenZ)] -
                    uCurr[_idx_kernel(i, j, k, lenX, lenY, lenZ)]
                );
            }
        }
    }
}

int main(int argc, char** argv) {
    std::cout << std::scientific << std::setprecision(6);
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &blocksTotalCount);
    MPI_Comm_rank(MPI_COMM_WORLD, &blockId);

    if (blockId == 0) {
        std::cin >> gridEqDim.x >> gridEqDim.y >> gridEqDim.z;
        std::cin >> blockEqDim.x >> blockEqDim.y >> blockEqDim.z;
        std::cin >> outFileName;
        std::cin >> epsilon;
        std::cin >> lX >> lY >> lZ;
        std::cin >> uDown >> uUp >> uLeft >> uRight >> uFront >> uBack >> u0;
        stringLen = outFileName.length();
    }
    MPI_Bcast(&gridEqDim.x, 1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(&gridEqDim.y, 1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(&gridEqDim.z, 1, MPI_INT, 0, MPI_COMM_WORLD);

    MPI_Bcast(&blockEqDim.x, 1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(&blockEqDim.y, 1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(&blockEqDim.z, 1, MPI_INT, 0, MPI_COMM_WORLD);

    MPI_Bcast(&epsilon, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

    MPI_Bcast(&lX, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(&lY, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(&lZ, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

    MPI_Bcast(&uDown, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(&uUp, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(&uLeft, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(&uRight, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(&uFront, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(&uBack, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

    MPI_Bcast(&u0, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    printGlobalVariableToCerr();
    castFileName();
    MPI_Barrier(MPI_COMM_WORLD);
    blockPos = getPositionByBlockId(blockId);

    int blockTotalSize = (blockEqDim.x + 2LL) * (blockEqDim.y + 2LL) * (blockEqDim.z + 2LL);
    int blockTotalSizeSmall = blockEqDim.x * blockEqDim.y * blockEqDim.z;

    double* uCurr = new double[blockTotalSize];
    double* diffs = new double[blockTotalSizeSmall];

    double hX = lX / (double)((long long)gridEqDim.x * blockEqDim.x);
    double hY = lY / (double)((long long)gridEqDim.y * blockEqDim.y);
    double hZ = lZ / (double)((long long)gridEqDim.z * blockEqDim.z);


    int device_count;
    hipGetDeviceCount(&device_count);
    hipSetDevice(blockId % device_count);

    for (int i = -1; i <= blockEqDim.x; i++) {
        for (int j = -1; j <= blockEqDim.y; j++) {
            for (int k = -1; k <= blockEqDim.z; k++) {
                uCurr[_idx(i, j, k)] = u0;
            }
        }
    }

    double max_diff = 0.0;

    double* dev_uCurr, *dev_uNext, *dev_diffs;
    CEH(hipMalloc(&dev_uCurr, sizeof(double) * blockTotalSize));
    CEH(hipMalloc(&dev_uNext, sizeof(double) * blockTotalSize));
    CEH(hipMalloc(&dev_diffs, sizeof(double) * blockTotalSize));

    do {
        MPI_Barrier(MPI_COMM_WORLD);

        reciveAndSendUpSide(uCurr);
        reciveAndSendDownSide(uCurr);
        reciveAndSendBackSide(uCurr);
        reciveAndSendFrontSide(uCurr);
        reciveAndSendLeftSide(uCurr);
        reciveAndSendRightSide(uCurr);


        CEH(hipMemcpy(dev_uCurr, uCurr, sizeof(double) * blockTotalSize, hipMemcpyHostToDevice));
        matrix_calc_kernel << < BLOCKS, THREADS >> > (dev_uCurr, dev_uNext, dev_diffs, blockEqDim.x, blockEqDim.y, blockEqDim.z, hX, hY, hZ);
        CEH(hipGetLastError());
        CEH(hipMemcpy(uCurr, dev_uNext, sizeof(double) * blockTotalSize, hipMemcpyDeviceToHost));

        thrust::device_ptr<double> begin = thrust::device_pointer_cast(dev_diffs);
        thrust::device_ptr<double> max_ptr;
        max_ptr = thrust::max_element(begin, begin + blockTotalSizeSmall);
        max_diff = *max_ptr;
        std::cerr << '[' << max_diff << ']' << '\n';
        //double* diffff = new double[blockTotalSizeSmall];
        //CEH(hipMemcpy(diffff, dev_diffs, blockTotalSizeSmall, hipMemcpyDeviceToHost));
        //printExclusiveBlock(uCurr);
        MPI_Barrier(MPI_COMM_WORLD);
        MPI_Allgather(&max_diff, 1, MPI_DOUBLE, diffs, 1, MPI_DOUBLE, MPI_COMM_WORLD);
        max_diff = 0.0;
        for (int i = 0; i < blocksTotalCount; ++i) {
            max_diff = std::max(max_diff, diffs[i]);
        }
    } while (max_diff > epsilon);
    MPI_Barrier(MPI_COMM_WORLD);
    multiPrint(uCurr);
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Finalize();
    hipFree(dev_uCurr);
    hipFree(dev_uNext);
    delete[] uCurr;
    delete[] diffs;
    return 0;
}
